#include "hip/hip_runtime.h"
/*
 * GPUNet.cpp
 *
 *  Created on: Jan 5, 2014
 *      Author: trevor
 */

#include "GPUNet.h"
#include "NetTrainer.h"
#include <boost/lexical_cast.hpp>
#include <boost/algorithm/string.hpp>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <thrust/device_vector.h>
#include "hiprand/hiprand_kernel.h"

/*
 * ------------ CUDA ------------
 */


/**
 * Get a random number within a given float range
 * \param min float
 * \param max float
 * \param i int
 * \param *global hiprandState
 */
__device__ float get_random_range(float min, float max, int i, hiprandState *global) {
	hiprandState local = global[i];
	float r = hiprand_uniform(&local);
	global[i] = local;
	return min + r * (max - min);
}

__device__ float get_ih_weight(float* ih_weights, int n_hidden, int i, int h) {
	return ih_weights[n_hidden*i + h];
}

__device__ float get_ho_weight(float* ho_weights, int n_output, int h, int o) {
	return ho_weights[n_output*h + o];
}

/**
 * Compute the sigmoid value of a given float
 * \param x the value to compute the sigmoid of
 */
__device__ inline float sigmoid(float x) {
	return 1.0 / (1.0 + exp(-x));
}


/**
 * Compute the output gradient given specific output and target values
 * \param output float
 * \param target float
 */
__device__ float calc_output_gradient(float output, float target) {
	return output * (1 - output) * (target - output);
}


/**
 * Clamp the output to 0 or 1 if within .1
 *\param f the value to clamp
 */
__device__ int clamp(float f) {
	if (f < .1) {
		return 0;
	} else if (f > .9) {
		return 1;
	} else {
		return -1;
	}
}

/*
 *
 * ------------- Initialization kernels ---------------
 *
 */


/**
 * Initialize random seeds in CUDA
 */
__global__ void curand_setup(hiprandState *state) {
	unsigned int seed = (unsigned int)clock64();
	int id = threadIdx.x;
	hiprand_init(seed, id, 0, &state[id]);
}

__global__ void curand_setup_v2(int n, hiprandState *state) {
	unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < n) {
		unsigned int seed = (unsigned int)clock64();
		hiprand_init(seed, id, 0, &state[id]);
	}
}

/**
 * initialize nodes to 0 or 1 if bias
 * block(1), threads(n_nodes+1)
 */
__global__ void init_nodes_layer(float *nodes) {
	int i = threadIdx.x;
	if (i == blockDim.x-1)
		nodes[i] = 1;
	else
		nodes[i] = 0;
}

__global__ void init_nodes_layer_v2(int n, float *nodes) {
	unsigned int i = blockIdx.x * blockDim.x+threadIdx.x;
	if (i < n) {
		if (i == n-1)
			nodes[i] = 1;
		else
			nodes[i] = 0;
	}
}

/**
 * block(1), threads(n_output)
 * set all output nodes to 0
 */
__global__ void init_nodes_output(float *output) {
	int i = threadIdx.x;
	output[i] = 0;
}

__global__ void init_nodes_output_v2(int n, float *output) {
	unsigned int i = blockIdx.x * blockDim.x+threadIdx.x;
	if (i < n) {
		output[i] = 0;
	}
}

//block(1), threads(n_layer1+1, n_layer2)
__global__ void init_weights(float *weights, hiprandState *state) {
	// r is the range for random values
	float r = 1.0 / sqrt((float)blockDim.x-1);

	int node_l1 = threadIdx.x;
	int node_l2 = threadIdx.y;
	weights[blockDim.y*node_l1 + node_l2] = get_random_range(-r, r, blockDim.y*node_l1 + node_l2, state);
}


__global__ void init_weights_v2(int n1, int n2, float *weights, hiprandState *state) {
	unsigned int i = blockIdx.x * blockDim.x+threadIdx.x;
	// r is the range for random values
	if (i < (n1+1)*n2) {
		float r = 1.0 / sqrt((float)blockDim.x-1);
		int node_l1 = i % (n1+1);
		int node_l2 = i % n2;
		weights[n2*node_l1 + node_l2] = get_random_range(-r, r, n2*node_l1 + node_l2, state);
	}
}



// block(1), threads(n_layer1+1, n_layer2)
__global__ void init_deltas(float *deltas) {
	int node_l1 = threadIdx.x;
	int node_l2 = threadIdx.y;

	deltas[blockDim.y*node_l1 + node_l2] = 0;
}

__global__ void init_deltas_v2(int n1, int n2, float *deltas) {
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < (n1+1)*n2) {
		int node_l1 = i % (n1+1);
		int node_l2 = i % n2;

		deltas[n2*node_l1 + node_l2] = 0;
	}
}




/*
 * --------------- Referencing and simple set function ---------------
 * set bias
 * set input/target[ref]
 *
 */

//used when copying patterns to device
__global__ void set_bias(int n_input, float *d_inp) {
	d_inp[n_input] = 1;
}


/*
 * -------------- Error calculation ---------------
 * output_correct
 * mse_sum
 *
 */

__device__ bool d_correct_result = true;

__device__ int d_num_correct = 0;
__device__ float d_acc = 0;
// called with blocks = (1), threads = (n_output)
// d_correct_result must be set to true before each call
// d_correct_result is copied back to host afterwards
__global__ void output_correct(float *output, float *target) {
	int i = threadIdx.x;
	if (d_correct_result && clamp(output[i]) != target[i]) {
		d_correct_result = false;
	}
}

__global__ void output_correct_v2(float *output, float *target, int n_output) {
	for (int i = 0; i < n_output; ++i) {
		if (clamp(output[i]) != clamp(target[i])) {
			return;
		}
	}
	++d_num_correct;
}

__global__ void calc_acc(int n_patterns) {
	d_acc = ((float)d_num_correct/n_patterns * 100);
	d_num_correct = 0;
}

__device__ float d_mse_sum = 0;
__device__ float d_mse = 0; //current mse

// called with blocks = (1), threads = (n_output)
// d_mse_sum must be set to 0 before each call
// d_mse_sum is copied back to host afterwards
__global__ void mse_sum(float *output, float *target) {
	int i = threadIdx.x;
	d_mse_sum += pow((output[i] - target[i]), 2);
}


/**
 * single threaded
 */
__global__ void mse_sum_v2(float *output, float *target, int n_output) {
	float sum = 0;
	for (int i = 0; i < n_output; ++i) {
		sum += pow(output[i] - target[i], 2);
	}
	d_mse_sum += sum;
}

/**
 * single threaded
 */
__global__ void calc_mse(int n_output, int n_patterns) {
	d_mse = d_mse_sum / (n_output * n_patterns);
	d_mse_sum = 0;
}



/*
 * ---- feed forward kernels -----------
 *
 * method 1 calculates each node in the next layer with a single thread computing for each output node
 * method 2 has a thread for each term in the linear combination to compute the output
 *     then the activation is computed after syncing threads.
 */

/*
 * to measure bandwidth:
 * (bytes read + bytes writen) / (time secs * 10^9) = gb
 *
 * bytes read = 4* ((n_layer1+1)*2),
 * bytes written = 4* (n_layer2)
 * total/thread = 4*((n_layer1+1)*2 + n_layer2)
 * threads l1 -> l2 = n_hidden
 * threads l2 -> l3 = n_output
 *
 * total_l1->l2 = n_hidden*4*((n_layer1+1)*2 + n_layer2)
 * total_l2->l3 = n_output*4*((n_layer2+1)*2 + n_layer3)
 *
 * total = total_l1->l2 + total_l2->l3;
 */
__global__ void feed_forward_layer_v1(int n_layer1, int n_layer2, float* layer1, float* layer2, float* weights) {
	int n = threadIdx.x; // node to compute;

	float r = 0;
	for (int i = 0; i <= n_layer1; ++i) { //include bias
		r += layer1[i] * weights[n_layer2*i + n];
	}
	layer2[n] = sigmoid(r);
}

/*
 * Generic version, called with pow of 2 threads
 */
__global__ void feed_forward_layer_v1_2(int n_layer1, int n_layer2, float* layer1, float* layer2, float* weights) {
	unsigned int n = blockIdx.x * blockDim.x+threadIdx.x; // node to compute;

	if (n < n_layer2) {
		float r = 0;
		for (int i = 0; i <= n_layer1; ++i) { //include bias
			r += layer1[i] * weights[n_layer2*i + n];
		}
		layer2[n] = sigmoid(r);
	}
}


/*
 * evoked with blocks(num nodes layer 2), threads(num nodes layer 1)
 *
 * sums holds the values of each term in the linear combination
 * n1t1, n1t2, ... n1tm, n2t1, n2t2, ... , n2tm, ...
 *
 *
 * bandwidth calculation:
 *
 * per thread,
 * 	bytes read: 4*2
 * 	bytes written: 4*1
 *
 * (n_input+1)*n_hidden threads
 *
 * Total: (n_input+1)*n_hidden*4*3
 */
__global__ void feed_forward_layer_v2(int n_layer1, int n_layer2, float* layer1, float* layer2, float* weights, float* sums) {
	int j = blockIdx.x; //the node in the next layer to compute
	int i = threadIdx.x + threadIdx.y * blockDim.x; //0;//the term in the linear combination to compute

	if (i > n_layer1) {
		return;
	}

	sums[(n_layer1+1)*j+i] = layer1[i] * weights[n_layer2*i + j];
}

__global__ void feed_forward_layer_v2_2(int n_layer1, int n_layer2, float* layer1, float* layer2, float* weights, float* sums) {

	unsigned int i = blockIdx.x * blockDim.x+threadIdx.x; // input node

	if (i <= n_layer1) {
		int j = i % n_layer2;
		sums[(n_layer1+1)*j+i] = layer1[i] * weights[n_layer2*i + j];
	}

}


__global__ void compute_activation(float* nodes, float *sums, int stride) {
	int i = threadIdx.x;
	nodes[i] = sigmoid(sums[i*stride]);
}

/*
 * generic version
 */
__global__ void compute_activation_v2(float* nodes, float *sums, int n_layer, int stride) {
	unsigned int i = blockIdx.x * blockDim.x+threadIdx.x; // input node

	if (i < n_layer)
		nodes[i] = sigmoid(sums[i*stride]);
}


/*
 * n_nodes is the number of nodes in the previous layer
 *
 * bandwidth calc:
 * 	bytes read: 4
 * 	bytes written: 4
 *
 */
__global__ void reduction(int n_nodes_prev, int n_nodes_next, int itr, float* sums) {
	//initialize dynamic shared memory for floor(n_nodes_prev/2.0)*n_nodes_next floats

	//get index
	int i = threadIdx.x;
	int index = i + blockIdx.x * blockDim.x; // * n_nodes_prev
	//if in correct term and not leftover
	if ((i % (int)powf(2,itr)) == 0 && ((i+1) % n_nodes_prev) != 0) {
		//printf("sums[%d] = %f, sums[%d+%d] = %f\n", i, sums[i], i, (int)powf(2, itr-1), sums[i+ (int)powf(2,itr-1)]);
		sums[index] += sums[index + (int)powf(2,itr-1)];
	}
}

/*
 * Shared memory reduction, handles threads more efficiently.
 */
__global__ void split_reduce(int n, int offset, float *g_idata, float *g_odata) {
	extern __shared__ float sdata[];

	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x + offset;

	if (i < n + offset) { // if in our range of data
		//printf("i = %d\n", i);
		sdata[tid] = g_idata[i];
		__syncthreads();

		for (unsigned int s = 1; s < blockDim.x; s *= 2) {
			int index = 2 * s * tid;
			if (index < blockDim.x && (index + s) < n) {
				//printf("sdata[%d] = %f,  sdata[%d] = %f\n", index, sdata[index], index+s, sdata[index+s]);
				sdata[index] += sdata[index + s];
			}
			__syncthreads();
		}
		if (tid == 0) g_odata[blockIdx.x+offset] = sdata[0];
	}
}


/*
 *
 *
 * ------------ backprop kernels ---------
 *
 *
 */

/*
 * called with threads(n_output)
 */
__global__ void output_error_gradients(float* output, float* target, float* output_err_gradients) {
	int i = threadIdx.x;
	output_err_gradients[i] = calc_output_gradient(output[i], target[i]);
	//printf("out_err_grad[%d] = %f, output = %f, target = %f\n", i, output_err_gradients[i], output[i], target[i]);
}

/*
 * called generically, pow of 2 threads
 */
__global__ void output_error_gradients_v2(float* output, float* target, float* output_err_gradients, int no) {
	unsigned int i = blockIdx.x * blockDim.x+threadIdx.x;

	if (i < no) {
		output_err_gradients[i] = calc_output_gradient(output[i], target[i]);
		//printf("out_err_grad[%d] = %f, output = %f, target = %f\n", i, output_err_gradients[i], output[i], target[i]);
	}
}

/*
 * called with threads = (nh+1, no, 1)
 */
__global__ void update_hidden_output_deltas(int no, float l_rate, float momentum,
		float* hidden, float* output_err_gradients, float* delta_ho) {

	int j = threadIdx.x; //hidden node
	int k = threadIdx.y; //output node

	//This probably doesn't improve much
	//I assume that what really happens is every node writes the same value into shared memory
	//so really every thread is still doing the work
	//__shared__ float out_err_grad_k; out_err_grad_k = output_err_gradients[k];
	//__syncthreads();

	delta_ho[no*j + k] = l_rate * hidden[j] * output_err_gradients[k] + momentum * delta_ho[no*j + k];
	//printf("delta_ho(%d, %d) = %f, l_rate = %f, hidden[%d] = %f, out_err_gradients[%d] = %f, momentum = %f\n",
	//		j, k, delta_ho[no*j+k], l_rate, j, hidden[j], k, output_err_gradients[k], momentum);

}


/*
 * called generically with power of 2 threads
 */
__global__ void update_hidden_output_deltas_v2(int nh, int no, float l_rate, float momentum,
		float* hidden, float* output_err_gradients, float* delta_ho) {

	unsigned int x = blockIdx.x * blockDim.x+threadIdx.x;

	if (x < (nh+1)*no) { // if in range
		//NOTE: this was my bug, had (x % nh) not (x % (nh+1))
		int j = x % (nh+1); //input node
		int k = x % no; //hidden node

		delta_ho[no*j + k] = l_rate * hidden[j] * output_err_gradients[k] + momentum * delta_ho[no*j + k];
		//printf("delta_ho(%d, %d) = %f, l_rate = %f, hidden[%d] = %f, out_err_gradients[%d] = %f, momentum = %f\n",
		//			j, k, delta_ho[no*j+k], l_rate, j, hidden[j], k, output_err_gradients[k], momentum);
	}
}


__device__ float calc_hidden_gradient(int j, int no, float* hidden, float* d_ho_weights, float* output_err_gradients) {
	//get sum of hidden->output weights * output error gradients
	float s = 0;
	for (int k = 0; k < no; ++k)
		s += d_ho_weights[j*no + k] * output_err_gradients[k];

	//return error gradient
	return hidden[j] * (1 - hidden[j]) * s;
}

/*
 * called with threads = (nh)
 */
__global__ void hidden_error_gradients(int no, float* hidden, float* d_ho_weights, float* hidden_err_gradients, float* output_err_gradients) {
	int j = threadIdx.x;
	hidden_err_gradients[j] = calc_hidden_gradient(j, no, hidden, d_ho_weights, output_err_gradients);
	//printf("hidden_err_grad[%d] = %f\n", j, hidden_err_gradients[j]);
}

/*
 * called generically, pow of 2 threads
 */
__global__ void hidden_error_gradients_v2(int nh, int no, float* hidden, float* d_ho_weights, float* hidden_err_gradients, float* output_err_gradients) {
	unsigned int j = blockIdx.x * blockDim.x+threadIdx.x;

	if (j < nh) { //NOTE: another bug, had (j < (nh+1)*no), only nh nodes need calculated
		hidden_err_gradients[j] = calc_hidden_gradient(j, no, hidden, d_ho_weights, output_err_gradients);
		//printf("hidden_err_grad[%d] = %f\n", j, hidden_err_gradients[j]);
	}
}


/*
 * called with blocks(no), threads(nh)
 *
 * reduce this list and then call calc gradients
 *
 * TODO: generalize this to pow 2 threads / blocks
 */
__global__ void hidden_error_gradients_v3(int no, float *sums, float *d_ho_weights, float *output_err_gradients) {
	int j = threadIdx.x;
	int k = blockIdx.x;

	sums[j*no + k] = d_ho_weights[j*no + k] * output_err_gradients[k];
}


/*
 * called with threads(nh)
 *
 * TODO: generalize this to pow 2 threads / blocks
 */
__global__ void calc_gradients(float *sums, float *hidden, float*hidden_err_gradients) {
	int i = threadIdx.x;
	hidden_err_gradients[i] = hidden[i] * (1 - hidden[i]) * sums[i*blockDim.x];
}


__global__ void update_input_hidden_deltas(int nh, float l_rate, float momentum,
		float* input, float* hidden_err_gradients, float* delta_ih) {

	int i = threadIdx.y; //input node
	int j = threadIdx.x; //hidden node

	delta_ih[nh*i + j] = l_rate * input[i] * hidden_err_gradients[j] + momentum * delta_ih[nh*i + j];

	//printf("delta_ho(%d, %d) = %f, l_rate = %f, input[%d] = %f, hidden_err_gradients[%d] = %f, momentum = %f\n",
	//			i, j, delta_ih[nh*i + j], l_rate, i, input[i], j, hidden_err_gradients[j], momentum);
}

/*
 * called with any number of blocks / threads
 * normally, 128 or other power of 2
 */
//TODO: perhaps there is a way to store the hidden_err_gradient[j] in shared memory
__global__ void update_input_hidden_deltas_v2(int ni, int nh, float l_rate, float momentum,
		float* input, float* hidden_err_gradients, float* delta_ih) {
	unsigned int x = blockIdx.x * blockDim.x+threadIdx.x;

	if (x < (ni+1)*nh) {
		int i = x % (ni+1); //input node, NOTE: same bug as before
		int j = x % nh; //hidden node

		delta_ih[nh*i + j] = l_rate * input[i] * hidden_err_gradients[j] + momentum * delta_ih[nh*i + j];

		//printf("delta_ih(%d, %d) = %f, l_rate = %f, input[%d] = %f, hidden_err_gradients[%d] = %f, momentum = %f\n",
		//			i, j, delta_ih[nh*i + j], l_rate, i, input[i], j, hidden_err_gradients[j], momentum);
	}
}



/*
 * weight update
 */

/*
 * called generically with power of 2 threads
 */
__global__ void update_weights_v2(int n1, int n2, float *d_weights, float *deltas) {
	unsigned int x = blockIdx.x * blockDim.x+threadIdx.x;

	if (x < (n1+1)*n2) {
		int i = x % (n1+1); //layer 1 node, NOTE: same bug
		int j = x % n2; //layer 2 node

		d_weights[i*n2 + j] += deltas[i*n2 + j];
		//printf("d_weights(%d, %d) = %f, deltas(%d, %d) = %f\n", i, j, d_weights[n2*i+j], i, j, deltas[n2*i + j]);
	}
}


//blocks(n_output), threads(n_hidden+1)
__global__ void update_weights_ho(int no, float* d_ho_weights, float* deltas_ho) {
	int k = blockIdx.x; //output
	int j = threadIdx.x; //hidden

	d_ho_weights[j*no + k] += deltas_ho[j*no + k];
}

//blocks(n_hidden), threads(n_input+1)
__global__ void update_weights_ih(int nh, float* d_ih_weights, float* deltas_ih) {
	int k = blockIdx.x; //hidden
	int j = threadIdx.x; //input

	d_ih_weights[j*nh + k] += deltas_ih[j*nh + k];
}

__global__ void print_gpu_net(int n_input, int n_hidden, int n_output,
		float *input, float *hidden, float *output, float *ih_weights, float *ho_weights) {
	for (int i = 0; i <= n_input; ++i) {
		printf("input %d: %f, ", i, input[i]);
	}
	printf("\n");
	for (int i = 0; i <= n_input; ++i) {
		for (int j = 0; j < n_hidden; ++j) {
			printf("ih weight (%d, %d): %f, ", i, j, ih_weights[n_hidden*i + j]);
		}
	}
	printf("\n");
	for (int i = 0; i <= n_hidden; ++i) {
		printf("hidden %d: %f, ", i, hidden[i]);
	}
	printf("\n");
	for (int i = 0; i <= n_hidden; ++i) {
		for (int j = 0; j < n_output; ++j) {
			printf("ho weight (%d, %d): %f, ", i, j, ho_weights[n_output*i + j]);
		}
	}
	printf("\n");
	for (int i = 0; i < n_output; ++i) {
		printf("output %d: %f, ", i, output[i]);
	}
	printf("\n");
}

/*
 *
 * --------- Debugging ------------
 *
 */

__global__ void print_floats(int n_input, float* d_input_2, FeatureVector *d_fv) {
	printf("d_fv.input: %f\n", d_fv->input[0]);
	printf("d_fv.input: %f\n", d_fv->input[1]);
	printf("d_fv.input: %f\n", d_fv->input[2]);
	for(int i = 0; i < n_input; ++i) {
		printf("%d: %f\n", i, d_input_2[i]);
	}
}

__global__ void print_floats2(int n_input, FeatureVector *d_fv) {
	printf("d_fv.input: %f\n", d_fv->input[0]);
	printf("d_fv.input: %f\n", d_fv->input[1]);
}

__global__ void print_all(int n_input, int n_output, FeatureVector **dv) {
	for (int i = 0; i < 4; ++i) {
		printf("Pattern %d\n", i);
		printf("Input: ");
		for (int j = 0; j < n_input; ++j) {
			printf("%f ", dv[i]->input[j]);
		}
		printf("\nTarget: ");
		for (int k = 0; k < n_output; ++k) {
			printf("%f ", dv[i]->target[k]);
		}
		printf("\n");
	}

}

__global__ void print_target(int n_output, float *target) {
	for (int i = 0; i < n_output; ++i) {
		printf("target[%d] = %f\n", i, target[i]);
	}
}

__global__ void print_input(int n_input, float *input) {
	for (int i = 0; i < n_input+1; i++) {
		printf("input[%d] = %f\n", i, input[i]);
	}
}

/*
 * ---------- Constructors -------------
 */

GPUNet::GPUNet() {
	n_input = 0;
	GPUNet::init_structure(0, 0, GPUNet::STANDARD);
	GPUNet::init_vars();
}

GPUNet::GPUNet(int ni, int no, GPUNet::NetworkStructure net_type) {
	n_input = 0;
	GPUNet::init_structure(ni, no, net_type);
	GPUNet::init_vars();
}

GPUNet::GPUNet(std::string net_file) {
	std::cout << "Initializing from net file: " << net_file << "." << std::endl;
	init_vars();
}

GPUNet::~GPUNet() {
	hipFree(d_input);
	hipFree(d_hidden);
	hipFree(d_output);
	hipFree(d_target);
	hipFree(d_ih_weights);
	hipFree(d_ho_weights);
	hipFree(d_ih_deltas);
	hipFree(d_ho_deltas);
	hipFree(d_hid_err_gradients);
	hipFree(d_out_err_gradients);

	delete[] h_output;
	delete[] gpu_mem;
}

/*
 * -------------- public ---------------
 */


void GPUNet::init_structure(int ni, int no, GPUNet::NetworkStructure net_type) {
	if (n_input != 0) { // constructor initializing nodes has been called, error out
		std::cerr << "Network has already been initialized" << std::endl;
	} else if (ni != 0) { // if not empty constructor
		n_input = ni;
		n_output = no;
		GPUNet::net_type = net_type;
		if (net_type == GPUNet::STANDARD) {
			n_hidden = ceil(2.0/3.0*ni);
		} else if (net_type == GPU_ARCH_OPT) {
			n_hidden = 128 / (2.0/3.0*ni) * 128;
		} else {
			std::cerr << "Invalid network type: " << net_type << std::endl;
			exit(1);
		}
	}
}

void GPUNet::init_vars() {
	max_epochs = GPU_MAX_EPOCHS;
	l_rate = GPU_LEARNING_RATE;
	momentum = GPU_MOMENTUM;
	desired_acc = GPU_DESIRED_ACCURACY;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&n_gpus));

	epoch = 0;
	trainingSetAccuracy = 0;
	validationSetAccuracy = 0;
	generalizationSetAccuracy = 0;
	trainingSetMSE = 0;
	validationSetMSE = 0;
	generalizationSetMSE = 0;

	start = 0;
	finish = 0;

	/*
	 * device
	 */
	d_input = NULL;
	d_hidden = NULL;
	d_output = NULL;
	d_target = NULL;

	d_ih_weights = NULL;
	d_ho_weights = NULL;

	d_ih_deltas = NULL;
	d_ho_deltas = NULL;

	d_hid_err_gradients = NULL;
	d_out_err_gradients = NULL;

	/*
	 * host validation
	 */
	h_output = new float[n_output];
	h_ih_weights = new float[(n_input+1)*n_hidden];
	h_ho_weights = new float[(n_hidden+1)*n_output];

	//init gpu mem to 0 for each gpu
	gpu_mem = new size_t[n_gpus];
	memset(gpu_mem, 0, n_gpus*sizeof(size_t));
	for (int i = 0; i < n_gpus; ++i) {
		gpu_mem[i] = 0;
	}
}


int GPUNet::get_next_int(std::ifstream &in) {
	std::string line;
	std::getline(in, line);
	std::vector<std::string> res;
	boost::split(res, line, boost::is_any_of("="));
	return boost::lexical_cast<int>(res[1]);
}

long GPUNet::get_next_long(std::ifstream &in) {
	std::string line;
	std::getline(in, line);
	std::vector<std::string> res;
	boost::split(res, line, boost::is_any_of("="));
	return boost::lexical_cast<long>(res[1]);
}

float GPUNet::get_next_float(std::ifstream &in) {
	std::string line;
	std::getline(in, line);
	std::vector<std::string> res;
	boost::split(res, line, boost::is_any_of("="));
	return boost::lexical_cast<float>(res[1]);
}

float* GPUNet::get_next_list(std::ifstream &in) {
	std::string line;
	std::getline(in, line);
	std::vector<std::string> res;
	boost::split(res, line, boost::is_any_of("="));
	std::vector<std::string> list;
	boost::split(list, res[1], boost::is_any_of(", "));

	float *fl_list = new float[list.size()];
	//just overwrite random GPU values
	for (size_t i = 0; i < list.size(); ++i) {
		fl_list[i] = boost::lexical_cast<float>(list[i]);
	}
	return fl_list;
}

bool GPUNet::read_net(std::string fname) {
	std::ifstream in(fname.c_str());
	if (in.is_open()) {
		// num epochs
		epoch = get_next_long(in);
		max_epochs = get_next_long(in);
		net_type = (GPUNet::NetworkStructure)get_next_int(in);

		//skip n_layers
		get_next_int(in);
		n_input = get_next_int(in);
		n_hidden = get_next_int(in);
		n_output = get_next_int(in);

		alloc_dev_mem();

		l_rate = get_next_float(in);
		momentum = get_next_float(in);
		desired_acc = get_next_float(in);
		trainingSetAccuracy = get_next_float(in);
		generalizationSetAccuracy = get_next_float(in);
		validationSetAccuracy = get_next_float(in);
		trainingSetMSE = get_next_float(in);
		generalizationSetMSE = get_next_float(in);
		validationSetMSE = get_next_float(in);

		float *ih_weights = get_next_list(in);
		CUDA_CHECK_RETURN(hipMemcpy(d_ih_weights, ih_weights, (n_input+1)*n_hidden*sizeof(float), hipMemcpyHostToDevice));
		float *ho_weights = get_next_list(in);
		CUDA_CHECK_RETURN(hipMemcpy(d_ho_weights, ho_weights, (n_hidden+1)*n_output*sizeof(float), hipMemcpyHostToDevice));

		delete[] ih_weights;
		delete[] ho_weights;
	} else {
		std::cout << "Could not read net file!" << std::endl;
		return false;
	}
	return true;
}

/*
 * allocate memory on device for
 * input, hidden, output, target
 * ih_weights, ho_weights
 * ih_deltas, ho_deltas
 * hid_err_gradients
 * out_err_gradients
 */

void GPUNet::alloc_dev_mem() {
	//nodes
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_input, (n_input+1)*sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_hidden, (n_hidden+1)*sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_output, (n_output)*sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_target, (n_output)*sizeof(float)));
	add_gpu_mem((n_input+n_hidden+(2*n_output)+2)*sizeof(float));

	//weights
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_ih_weights, ((n_input+1)*n_hidden)*sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_ho_weights, ((n_hidden+1)*n_output)*sizeof(float)));
	add_gpu_mem(((n_input+1)*n_hidden + (n_hidden+1)*n_output)*sizeof(float));

	//create delta arrays, include bias
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_ih_deltas, ((n_input+1)*n_hidden)*sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_ho_deltas, ((n_hidden+1)*n_output)*sizeof(float)));
	add_gpu_mem(((n_input+1)*n_hidden + (n_hidden+1)*n_output)*sizeof(float));

	//error gradients
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_hid_err_gradients, (n_hidden+1)*sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_out_err_gradients, (n_output+1)*sizeof(float)));
	add_gpu_mem((n_hidden + n_output + 2)*sizeof(float));

	std::cout << "Memory allocated on device." << std::endl;
}

/*
 * Note: assumes sizes of networks are the same
 * This is for testing purposes so that
 * I can have identical networks.
 */
void GPUNet::init_from_net(Net &net, NetData &d) {
	//copy first pattern to input neurons so it is copied to device, instead of zeros
	for (int i = 0; i < net.n_input; ++i) {
		net.inputNeurons[i] = d.get_training_dataset()->training_set[0]->input[i];
	}

	// so hidden and output initialized to 0
	CUDA_CHECK_RETURN(hipMemcpy(d_input, net.inputNeurons, (net.n_input)*sizeof(float), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_hidden, net.hiddenNeurons, (net.n_hidden)*sizeof(float), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_output, net.outputNeurons, (net.n_output)*sizeof(float), hipMemcpyHostToDevice));

	set_bias<<<1,1>>>(n_input, d_input);
	set_bias<<<1,1>>>(n_hidden, d_hidden);

	CUDA_CHECK_RETURN(hipMemcpy(d_ih_weights, net.wInputHidden, (net.n_input+1)*(net.n_hidden)*sizeof(float), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_ho_weights, net.wHiddenOutput, (net.n_hidden+1)*(net.n_output)*sizeof(float), hipMemcpyHostToDevice));

	//init deltas to 0
	dim3 ih_threads(n_input+1, n_hidden);
	dim3 ho_threads(n_hidden+1, n_output);
	init_deltas<<<1, ih_threads>>>(d_ih_deltas);
	init_deltas<<<1, ho_threads>>>(d_ho_deltas);

	std::cout << "data copied to device\n\n";
}


void GPUNet::init_net() {
	int threads = 128;

	//init nodes to all 0
	init_nodes_layer_v2<<<(n_input+1+threads-1)/threads, threads>>>(n_input+1, d_input);
	init_nodes_layer_v2<<<(n_hidden+1+threads-1)/threads, threads>>>(n_hidden+1, d_hidden);
	init_nodes_output_v2<<<(n_output+threads-1)/threads, threads>>>(n_output, d_output);

	//init weights to random vals
	hiprandState *state;
	std::cout << "size of hiprandState: " << sizeof(hiprandState) << std::endl;
	CUDA_CHECK_RETURN(hipMalloc(&state, (n_input+1)*n_hidden*sizeof(hiprandState)));
	curand_setup<<<1, (n_input+1)*n_hidden>>>(state);
	//curand_setup_v2<<<((n_input+1)*n_hidden+threads-1)/threads, threads>>>((n_input+1)*n_hidden, state);

	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	dim3 ih_threads(n_input+1, n_hidden);
	init_weights<<<1, ih_threads>>>(d_ih_weights, state);
	//init_weights_v2<<<((n_input+1)*n_hidden+threads-1)/threads, threads>>>(n_input+1, n_hidden, d_ih_weights, state);
	CUDA_CHECK_RETURN(hipFree(state));

	CUDA_CHECK_RETURN(hipMalloc(&state, (n_hidden+1)*n_output*sizeof(hiprandState)));
	curand_setup<<<1, (n_hidden+1)*n_output>>>(state);
	//curand_setup_v2<<<((n_hidden+1)*n_output+threads-1)/threads, threads>>>((n_hidden+1)*n_output, state);

	dim3 ho_threads(n_hidden+1, n_output);
	init_weights<<<1, ho_threads>>>(d_ho_weights, state);
	//init_weights_v2<<<((n_hidden+1)*n_output+threads-1)/threads, threads>>>(n_hidden+1, n_output, d_ho_weights, state);
	CUDA_CHECK_RETURN(hipFree(state));

	//init deltas to 0
	init_deltas<<<1, ih_threads>>>(d_ih_deltas);
	//init_deltas_v2<<<((n_input+1)*n_hidden+threads-1)/threads, threads>>>(n_input+1, n_hidden, d_ih_deltas);
	init_deltas<<<1, ho_threads>>>(d_ho_deltas);
	//init_deltas_v2<<<((n_hidden+1)*n_output+threads-1)/threads, threads>>>(n_hidden+1, n_output, d_ho_deltas);

	std::cout << "net initialized" << std::endl;
}

void GPUNet::set_learning_rate(float lr) {
	l_rate = lr;
}

void GPUNet::set_momentum(float m) {
	momentum = m;
}

void GPUNet::set_training_params(float lr, float m) {
	l_rate = lr;
	momentum = m;
}

void GPUNet::set_max_epochs(int me) {
	max_epochs = me;
}

void GPUNet::set_desired_accuracy(float acc) {
	desired_acc = acc;
}

void GPUNet::set_stopping_conds(int me, float acc) {
	max_epochs = me;
	desired_acc = acc;
}


/*
 * to keep it simple, run in 1 thread
 */
void GPUNet::print_net() {
	print_gpu_net<<<1, 1>>>(n_input, n_hidden, n_output,
			d_input, d_hidden, d_output, d_ih_weights, d_ho_weights);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
}


/*
 * transfer weights back to host
 * write important data (num_epochs, layers, nodes/layer, l_rate, momentum, max_epochs, desired_acc, current mse, current acc)
 *
 */
void GPUNet::write_net(std::string fname) {
	std::ofstream of(fname.c_str());

	float *ih_weights = new float[(n_input+1)*(n_hidden)];
	float *ho_weights = new float[(n_hidden+1)*(n_output)];

	CUDA_CHECK_RETURN(hipMemcpy(ih_weights, d_ih_weights, (n_input+1)*(n_hidden)*sizeof(float), hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipMemcpy(ho_weights, d_ho_weights, (n_hidden+1)*(n_output)*sizeof(float), hipMemcpyDeviceToHost));

	if (of.is_open()) {
		of << "num_epochs=" << epoch << "\n";
		of << "max_epochs=" << max_epochs << "\n";
		of << "net_type=" << net_type << "\n";
		of << "num_layers=" << 3 << "\n";
		of << "n_layer_0=" << n_input << "\n";
		of << "n_layer_1=" << n_hidden << "\n";
		of << "n_layer_2=" << n_output << "\n";
		of << "l_rate=" << l_rate << "\n";
		of << "momentum=" << momentum << "\n";
		of << "desired_acc=" << desired_acc << "\n";
		of << "tset_acc=" << trainingSetAccuracy << "\n";
		of << "gset_acc=" << generalizationSetAccuracy << "\n";
		of << "vset_acc=" << validationSetAccuracy << "\n";
		of << "tset_mse=" << trainingSetMSE << "\n";
		of << "gset_mse=" << generalizationSetMSE << "\n";
		of << "vset_mse=" << validationSetMSE << "\n";
		of << "weights_ih=";
		for (int i = 0, l = (n_input+1)*n_hidden; i < l; ++i) {
			of << ih_weights[i];
			if (i != l-1)
				of << ",";
		}
		of << "\n";
		of << "weights_ho=";
		for (int i = 0, l = (n_hidden+1)*n_output; i < l; ++i) {
			of << ho_weights[i];
			if (i != l-1)
				of << ",";
		}

		of.flush();
		of.close();
	} else {
		std::cout << "Could not write file!" << std::endl;
	}

	delete[] ih_weights;
	delete[] ho_weights;
}


int GPUNet::get_num_input() {
	return n_input;
}

int GPUNet::get_num_hidden() {
	return n_hidden;
}

int GPUNet::get_num_output() {
	return n_output;
}

int GPUNet::num_patterns_copyable(TrainingDataSet *tset) {
	//num patterns = integer div of available memory / mem for single pattern
	int bytes_per_pattern = sizeof(float)*((n_input+1)+(n_output));
	int cur_dev = get_current_device();
	int available_mem = total_dev_mem(cur_dev) - current_mem_usage(cur_dev);
	return available_mem / bytes_per_pattern;
}

void GPUNet::calc_dataset_parameters(TrainingDataSet *tset) {
	// calc num patterns copyable
	// num patterns = integer div of available memory / mem for single pattern
	int bytes_per_pattern = sizeof(float)*((n_input+1)+(n_output));
	int cur_dev = get_current_device();
	std::cout << "bytes per pattern="<<bytes_per_pattern<<std::endl;
	std::cout << "total dev mem="<< total_dev_mem(cur_dev)<<std::endl;
	std::cout << "current mem usage="<< current_mem_usage(cur_dev)<<std::endl;
	int available_mem = total_dev_mem(cur_dev) - current_mem_usage(cur_dev);
	std::cout << "available mem="<<available_mem<<std::endl;
	std::cout << "tset.size="<<tset->size()<<std::endl;
	n_copyable_patterns = available_mem / bytes_per_pattern;
	if (n_copyable_patterns > tset->size()) {
		n_copyable_patterns = tset->size();
	}
	// calc num sections
	// num_sections = ceil ( n_patterns / n_copyable_patterns)
	n_sections = (tset->size() + n_copyable_patterns - 1) / n_copyable_patterns;

	std::cout << "n_copyable_patterns="<<n_copyable_patterns<<", n_sections="<<n_sections<<std::endl;
}

void GPUNet::train_net(TrainingDataSet *tset) {
	std::cout << std::endl << "Neural Network Training Starting: " << std::endl
			<< "----------------------------------------------------" << std::endl
			<< "LR: " << l_rate << ", Momentum: " << momentum << ", Max Epochs: " << max_epochs << std::endl
			<< n_input << " Input Neurons, " << n_hidden << " Hidden Neurons, " << n_output << " Output Neurons" << std::endl
			<< "----------------------------------------------------" << std::endl << std::endl;

	FeatureVector** d_training_set;
	FeatureVector** d_generalization_set;
	FeatureVector** d_validation_set;
	if (num_patterns_copyable(tset) >= tset->size()) {
		//copy all patterns
		start = clock();
		copy_to_device_host_array_ptrs_biased(tset->training_set, &d_training_set);
		copy_to_device_host_array_ptrs_biased(tset->generalization_set, &d_generalization_set);
		copy_to_device_host_array_ptrs_biased(tset->validation_set, &d_validation_set);
		finish = clock();
		std::cout << "Copying entire dataset to device: " << ((float)finish-start)/CLOCKS_PER_SEC << std::endl;
	} else {
		//TODO: what do I do?
		// Copy as many as possible
		//
		// Should this be done in a separate thread
	}

	epoch = 0;
	//train network using training dataset for training and generalization dataset for testing
	//while ((trainingSetAccuracy < desired_acc) && epoch < max_epochs) {
	while (epoch < max_epochs) {
		//store previous accuracy
		//float previousTAccuracy = trainingSetAccuracy;
		//float previousGAccuracy = generalizationSetAccuracy;

		//use training set to train network
		//run_training_epoch(tset->training_set);
		//std::cout << "Calling run_training_epoch_dev" << std::endl;
		run_training_epoch_dev(d_training_set, tset->training_set.size());


		//get generalization set accuracy and MSE
		//get_set_accuracy_mse(tset->generalization_set, &generalizationSetAccuracy, &generalizationSetMSE);
		//get_set_accuracy_mse_dev(d_generalization_set, tset->generalization_set.size(), &generalizationSetAccuracy, &generalizationSetMSE);

		//print out change in training /generalization accuracy (only if a change is greater than a percent)
		//if (ceil(previousTAccuracy) != ceil(trainingSetAccuracy) || ceil(previousGAccuracy) != ceil(generalizationSetAccuracy)) {
			std::cout << "Epoch: " << epoch << std::endl;
			//std::cout << "; Test Set Acc:" << trainingSetAccuracy << "%, MSE: " << trainingSetMSE;
			//std::cout << ";\tGSet Acc:" << generalizationSetAccuracy << "%, MSE: " << generalizationSetMSE << std::endl;
		//}


		//previousTAccuracy = trainingSetAccuracy;
		//previousGAccuracy = generalizationSetAccuracy;

		//once training set is complete increment epoch
		++epoch;
	}

	//get validation set accuracy and MSE
	//get_set_accuracy_mse(tset->validation_set, &validationSetAccuracy, &validationSetMSE);
	//get_set_accuracy_mse_dev(d_validation_set, tset->validation_set.size(), &validationSetAccuracy, &validationSetMSE);

	//out validation accuracy and MSE
	std::cout << std::endl << "Training Complete. Elapsed Epochs: " << epoch << std::endl;
	//std::cout << "\tValidation Set Accuracy: " << validationSetAccuracy << std::endl;
	//std::cout << "\tValidation Set MSE: " << validationSetMSE << std::endl << std::endl;

	//free training set
	for (int i = 0; i < tset->training_set.size(); ++i) {
		CUDA_CHECK_RETURN(hipFree(d_training_set[i]->input));
		CUDA_CHECK_RETURN(hipFree(d_training_set[i]->target));
		free(d_training_set[i]);
	}
	free(d_training_set);
}



void GPUNet::train_net_sectioned(TrainingDataSet *tset) {
	std::cout << std::endl << "Neural Network Training Starting: " << std::endl
			<< "----------------------------------------------------" << std::endl
			<< "LR: " << l_rate << ", Momentum: " << momentum << ", Max Epochs: " << max_epochs << std::endl
			<< n_input << " Input Neurons, " << n_hidden << " Hidden Neurons, " << n_output << " Output Neurons" << std::endl
			<< "----------------------------------------------------" << std::endl << std::endl;

	calc_dataset_parameters(tset);
	epoch = 0;
	FeatureVector** d_training_set;

	if (n_sections == 1) { // no section copying necessary
		copy_to_device_host_array_ptrs_biased(tset->training_set, &d_training_set);
		std::cout << "data copied" << std::endl;
		while (epoch < max_epochs) {
			run_training_epoch_dev(d_training_set, tset->training_set.size());
			std::cout << "Epoch: " << epoch << std::endl;
			++epoch;
		}
	} else {
		while (epoch < max_epochs) {
			//copy a section and run partial epoch
			for (int i = 0; i < n_sections; ++i) {
				//copy patterns from [n_sections*n_patterns_copyable, (n_sections+1)*n_patterns_copyable)
				int p_start = i * n_copyable_patterns;
				int p_end = p_start + n_copyable_patterns;
				if (p_end > tset->training_set.size()) p_end = tset->training_set.size();
				std::cout << "copying section="<<i<<", pstart="<< p_start << ", pend="<<p_end << std::endl;
				copy_to_device_host_array_ptrs_biased_section(tset->training_set, &d_training_set, p_start, p_end, i == 0 && epoch == 0);
				std::cout << "data copied" << std::endl;
				run_training_epoch_dev(d_training_set, p_end-p_start);
			}

			std::cout << "Epoch: " << epoch << std::endl;
			//once training set is complete increment epoch
			++epoch;
		}
	}

	//out validation accuracy and MSE
	std::cout << std::endl << "Training Complete. Elapsed Epochs: " << epoch << std::endl;

	CUDA_CHECK_RETURN(hipMemcpyFromSymbol(&trainingSetMSE, HIP_SYMBOL(d_mse), sizeof(float), 0, hipMemcpyDeviceToHost));
	std::cout << "MSE = " << trainingSetMSE << std::endl;
	CUDA_CHECK_RETURN(hipMemcpyFromSymbol(&trainingSetAccuracy, HIP_SYMBOL(d_acc), sizeof(float), 0, hipMemcpyDeviceToHost));
	std::cout << "ACC = " << trainingSetAccuracy << std::endl;

	//free training set
	for (int i = 0; i < tset->training_set.size(); ++i) {
		CUDA_CHECK_RETURN(hipFree(d_training_set[i]->input));
		CUDA_CHECK_RETURN(hipFree(d_training_set[i]->target));
		free(d_training_set[i]);
	}
	free(d_training_set);
}

void GPUNet::get_set_accuracy_mse(thrust::host_vector<FeatureVector*> set, float* s_acc, float* s_mse) {
	int incorrect_patterns = 0;
	float mse = 0, mse_tmp = 0;
	bool correct_result = true;

	//TODO: copy multiple patters at once so bandwidth is not a limiting factor
	for (unsigned int i = 0; i < set.size(); ++i) {
		mse_tmp = 0;
		correct_result = true;

		//copy pattern to input
		CUDA_CHECK_RETURN(hipMemcpy(d_input, set[i]->input, (n_input+1)*sizeof(float), hipMemcpyHostToDevice));

		//copy target to dev
		CUDA_CHECK_RETURN(hipMemcpy(d_target, set[i]->target, (n_output)*sizeof(float), hipMemcpyHostToDevice));

		//feed forward input
		feed_forward_v1();

		CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(d_correct_result), &correct_result, sizeof(correct_result), 0, hipMemcpyHostToDevice));
		CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(d_mse_sum), &mse_tmp, sizeof(mse_tmp), 0, hipMemcpyHostToDevice));
		output_correct<<<1, n_output>>>(d_output, d_target);
		mse_sum<<<1, n_output>>>(d_output, d_target);
		CUDA_CHECK_RETURN(hipMemcpyFromSymbol(&correct_result, HIP_SYMBOL(d_correct_result), sizeof(correct_result), 0, hipMemcpyDeviceToHost));
		CUDA_CHECK_RETURN(hipMemcpyFromSymbol(&mse_tmp, HIP_SYMBOL(d_mse_sum), sizeof(mse_tmp), 0, hipMemcpyDeviceToHost));
		if (!correct_result)
			++incorrect_patterns;
		mse += mse_tmp;
	}

	*s_acc = 100 - ((float)incorrect_patterns/set.size() * 100);
	*s_mse = mse / (n_output * set.size());
}

void GPUNet::run_training_epoch(thrust::host_vector<FeatureVector*> feature_vecs) {
	print_net();
	int incorrect_patterns = 0;
	float mse = 0, mse_tmp = 0;
	bool correct_result = true;

	for (unsigned int i = 0; i < feature_vecs.size(); ++i) {
		mse_tmp = 0;
		correct_result = true;

		//copy pattern to input
		CUDA_CHECK_RETURN(hipMemcpy(d_input, feature_vecs[i]->input, (n_input+1)*sizeof(float), hipMemcpyHostToDevice));

		//copy target to dev
		CUDA_CHECK_RETURN(hipMemcpy(d_target, feature_vecs[i]->target, (n_output)*sizeof(float), hipMemcpyHostToDevice));

		//feed forward input
		feed_forward_v1();

		CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(d_correct_result), &correct_result, sizeof(correct_result), 0, hipMemcpyHostToDevice));
		CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(d_mse_sum), &mse_tmp, sizeof(mse_tmp), 0, hipMemcpyHostToDevice));
		output_correct<<<1, n_output>>>(d_output, d_target);
		mse_sum<<<1, n_output>>>(d_output, d_target);
		CUDA_CHECK_RETURN(hipMemcpyFromSymbol(&correct_result, HIP_SYMBOL(d_correct_result), sizeof(correct_result), 0, hipMemcpyDeviceToHost));
		CUDA_CHECK_RETURN(hipMemcpyFromSymbol(&mse_tmp, HIP_SYMBOL(d_mse_sum), sizeof(mse_tmp), 0, hipMemcpyDeviceToHost));
		if (!correct_result)
			++incorrect_patterns;
		mse += mse_tmp;

		//std::cout << "Correct result: " << correct_result << ", mse_tmp: " << mse_tmp << std::endl;
		//backprop target
		backprop_v1();
	}

	//std::cout << "MSE sum: " << mse << std::endl;
	//std::cout << "inc patterns: " << incorrect_patterns << std::endl;
	//update training accuracy and MSE
	trainingSetAccuracy = 100 - ((float)incorrect_patterns/feature_vecs.size() * 100);
	trainingSetMSE = mse / (n_output * feature_vecs.size());
}

void GPUNet::get_set_accuracy_mse_dev(FeatureVector **feature_vecs, size_t n_features, float* s_acc, float* s_mse) {
	int incorrect_patterns = 0;
	float mse = 0, mse_tmp = 0;
	bool correct_result = true;

	//TODO: copy multiple patters at once so bandwidth is not a limiting factor
	for (unsigned int i = 0; i < n_features; ++i) {
		mse_tmp = 0;
		correct_result = true;

		//feed forward input
		feed_forward_v1_2(feature_vecs[i]->input);

		CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(d_correct_result), &correct_result, sizeof(correct_result), 0, hipMemcpyHostToDevice));
		CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(d_mse_sum), &mse_tmp, sizeof(mse_tmp), 0, hipMemcpyHostToDevice));
		output_correct<<<1, n_output>>>(d_output, feature_vecs[i]->target);
		mse_sum<<<1, n_output>>>(d_output, feature_vecs[i]->target);
		CUDA_CHECK_RETURN(hipMemcpyFromSymbol(&correct_result, HIP_SYMBOL(d_correct_result), sizeof(correct_result), 0, hipMemcpyDeviceToHost));
		CUDA_CHECK_RETURN(hipMemcpyFromSymbol(&mse_tmp, HIP_SYMBOL(d_mse_sum), sizeof(mse_tmp), 0, hipMemcpyDeviceToHost));
		if (!correct_result)
			++incorrect_patterns;
		mse += mse_tmp;
	}

	*s_acc = 100 - ((float)incorrect_patterns/n_features * 100);
	*s_mse = mse / (n_output * n_features);
}

void GPUNet::run_training_epoch_dev(FeatureVector **feature_vecs, size_t n_features) {
	for (size_t i = 0; i < n_features; ++i) {
		feed_forward_v1_2(feature_vecs[i]->input);
		backprop_v2(feature_vecs[i]->input, feature_vecs[i]->target);
		CUDA_CHECK_RETURN(hipDeviceSynchronize());
	}
	calc_mse<<<1, 1>>>(n_output, n_features);
	calc_acc<<<1, 1>>>(n_features);
	//CUDA_CHECK_RETURN(hipDeviceSynchronize());
	//float mse = 0;
	//CUDA_CHECK_RETURN(hipMemcpyFromSymbol(&mse, HIP_SYMBOL(d_mse), sizeof(float), 0, hipMemcpyDeviceToHost));
	//std::cout << "Current mse = " << mse << std::endl;
}

/*
 * Reduce sums from len to n sums. Assumes len is a multiple of n.
 */
float* GPUNet::reduce(int n, int len, float* d_sums, float *d_y) {
	int step = len / n;

	float *res;
	for (int i = n-1; i >= 0; --i) {
		res = execute_split_reduction(step, i*step, d_sums, d_y);
	}
	return res;
}

/*
 * n is number of elements to sum
 * offset is where to start in the list
 * d_x is original list
 */
float* GPUNet::execute_split_reduction(int n, int offset, float *d_x, float *d_y) {
	bool result_in_y = false;
	int threads = 128;
	int blocks = (n+threads-1);

	if (n >= threads) {
		do {
			blocks /= threads;
			if (result_in_y)
				split_reduce<<<blocks, threads, threads*sizeof(float)>>>(n, offset, d_y, d_x);
			else
				split_reduce<<<blocks, threads, threads*sizeof(float)>>>(n, offset, d_x, d_y);
			result_in_y = !result_in_y;
			CUDA_CHECK_RETURN(hipDeviceSynchronize());
		} while (blocks/threads >= threads);
		if (result_in_y)
			//reduce0<<<1, threads, threads*sizeof(float)>>>(blocks, d_y, d_x);
			split_reduce<<<1, blocks, blocks*sizeof(float)>>>(n, offset, d_y, d_x);
		else
			//reduce0<<<1, threads, threads*sizeof(float)>>>(blocks, d_x, d_y);
			split_reduce<<<1, blocks, blocks*sizeof(float)>>>(n, offset, d_x, d_y);
		result_in_y = !result_in_y;
		CUDA_CHECK_RETURN(hipDeviceSynchronize());
	} else {
		split_reduce<<<1, n, n*sizeof(float)>>>(n, offset, d_x, d_y);
		result_in_y = !result_in_y;
	}

	if (result_in_y) {
		return d_y;
	} else {
		return d_x;
	}
}



void GPUNet::backprop_v1() {
	output_error_gradients<<<1, n_output>>>(d_output, d_target, d_out_err_gradients);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	/*
	 * called with threads = (nh+1, no, 1)
	 */
	dim3 hid_out_deltas(n_hidden+1, n_output);
	update_hidden_output_deltas<<<1, hid_out_deltas>>>(n_output, l_rate, momentum, d_hidden, d_out_err_gradients, d_ho_deltas);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	/*
	 * called with threads = (nh)
	 */
	hidden_error_gradients<<<1, n_hidden>>>(n_output, d_hidden, d_ho_weights,
			d_hid_err_gradients, d_out_err_gradients);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	update_weights_ho<<<n_output, n_hidden+1>>>(n_output, d_ho_weights, d_ho_deltas);

	dim3 in_hid_deltas(n_input+1, n_hidden);
	update_input_hidden_deltas<<<1, in_hid_deltas>>>(n_hidden, l_rate, momentum,
			d_input, d_hid_err_gradients, d_ih_deltas);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	update_weights_ih<<<n_hidden, n_input+1>>>(n_hidden, d_ih_weights, d_ih_deltas);
}



void GPUNet::backprop_v2(float *d_inp, float *d_tar) {
	hipStream_t mse_sum_stream, output_correct_stream, bprop_stream;
	CUDA_CHECK_RETURN(hipStreamCreate(&mse_sum_stream));
	CUDA_CHECK_RETURN(hipStreamCreate(&output_correct_stream));
	CUDA_CHECK_RETURN(hipStreamCreate(&bprop_stream));
	int n_threads = 128;

	//maintain mse state
	mse_sum_v2<<<1, 1, 0, mse_sum_stream>>>(d_output, d_tar, n_output);
	output_correct_v2<<<1, 1, 0, output_correct_stream>>>(d_output, d_tar, n_output);
	//CUDA_CHECK_RETURN(hipDeviceSynchronize());
	//float mse_sum = 0;
	//CUDA_CHECK_RETURN(hipMemcpyFromSymbol(&mse_sum, HIP_SYMBOL(d_mse_sum), sizeof(float), 0, hipMemcpyDeviceToHost));
	//std::cout << "Current mse_sum = " << mse_sum << std::endl;

	output_error_gradients_v2<<<(n_output+n_threads-1)/n_threads, n_threads, 0, bprop_stream>>>(d_output, d_tar, d_out_err_gradients, n_output);
	//CUDA_CHECK_RETURN(hipDeviceSynchronize());

	update_hidden_output_deltas_v2<<<((n_output*(n_hidden+1))+n_threads-1)/n_threads, n_threads, 0, bprop_stream>>>(n_hidden, n_output, l_rate, momentum, d_hidden, d_out_err_gradients, d_ho_deltas);
	//CUDA_CHECK_RETURN(hipDeviceSynchronize());

	hidden_error_gradients_v2<<<(n_hidden+n_threads-1)/n_threads, n_threads, 0, bprop_stream>>>(n_hidden, n_output, d_hidden, d_ho_weights,
			d_hid_err_gradients, d_out_err_gradients);
	//CUDA_CHECK_RETURN(hipDeviceSynchronize());

	update_weights_v2<<<((n_output*(n_hidden+1))+n_threads-1)/n_threads, n_threads, 0, bprop_stream>>>(n_hidden, n_output, d_ho_weights, d_ho_deltas);

	update_input_hidden_deltas_v2<<<((n_hidden*(n_input+1))+n_threads-1)/n_threads, n_threads, 0, bprop_stream>>>(n_input, n_hidden, l_rate, momentum,
				d_inp, d_hid_err_gradients, d_ih_deltas);
	//CUDA_CHECK_RETURN(hipDeviceSynchronize());

	update_weights_v2<<<((n_hidden*(n_input+1))+n_threads-1)/n_threads, n_threads, 0, bprop_stream>>>(n_input, n_hidden, d_ih_weights, d_ih_deltas);
}

void GPUNet::feed_forward_v1() {
	feed_forward_layer_v1<<<1, n_hidden>>>(n_input, n_hidden, d_input, d_hidden, d_ih_weights);
	// must finish previous layer before computing next
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	feed_forward_layer_v1<<<1, n_output>>>(n_hidden, n_output, d_hidden, d_output, d_ho_weights);

	//sync before measuring time
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
}

void GPUNet::feed_forward_v1_2(float *d_inp) {
	int threads = 128;
	feed_forward_layer_v1_2<<<(n_hidden+threads-1)/threads, threads>>>(n_input, n_hidden, d_inp, d_hidden, d_ih_weights);
	// must finish previous layer before computing next
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	feed_forward_layer_v1_2<<<(n_output+threads-1)/threads, threads>>>(n_hidden, n_output, d_hidden, d_output, d_ho_weights);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
}


void GPUNet::feed_forward_v2() {
	dim3 gridm2l1(n_hidden);
	dim3 threadsm2l1 = get_threadsm2l1();
	//std::cout << "threads layer 1: (" << threadsm2l1.x << " " << threadsm2l1.y << " " << threadsm2l1.z << ")" << std::endl;

	//float *a = new float[(n_input+1)*n_hidden];
	float* d_sums_l1, *d_y;
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_sums_l1, (n_input+1)*n_hidden*sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_y, (n_input+1)*n_hidden*sizeof(float)));

	feed_forward_layer_v2<<<gridm2l1, threadsm2l1>>>(n_input, n_hidden, d_input, d_hidden, d_ih_weights, d_sums_l1);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	d_sums_l1 = reduce(n_hidden, (n_input+1)*n_hidden, d_sums_l1, d_y);

	compute_activation<<<1, n_hidden>>>(d_hidden, d_sums_l1, n_input+1);

	// must finish previous layer before computing next
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	CUDA_CHECK_RETURN(hipFree(d_sums_l1));
	//GPUNet::add_gpu_mem(-(n_input+1)*n_hidden*sizeof(float));

	//grid size must be >= # nodes in next layer
	dim3 gridm2l2(n_output);
	//1 thread per grid
	dim3 threadsm2l2 = get_threadsm2l2();
	//std::cout << "threads layer 2: (" << threadsm2l2.x << " " << threadsm2l2.y << " " << threadsm2l2.z << ")" << std::endl;

	float *d_sums_l2;
	CUDA_CHECK_RETURN(hipMalloc(&d_sums_l2, n_hidden*n_output*sizeof(float)));
	CUDA_CHECK_RETURN(hipMemset(d_y, 0, (n_input+1)*n_hidden*sizeof(float)));
	//GPUNet::add_gpu_mem((n_hidden+1)*n_output*sizeof(float));

	feed_forward_layer_v2<<<gridm2l2, threadsm2l2>>>(n_hidden, n_output, d_hidden, d_output, d_ho_weights, d_sums_l2);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	d_sums_l2 = reduce(n_output, (n_hidden+1)*n_output, d_sums_l2, d_y);

	compute_activation<<<1, n_output>>>(d_output, d_sums_l2, n_hidden+1);

	//sync before measuring time
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	CUDA_CHECK_RETURN(hipFree(d_sums_l2));
	CUDA_CHECK_RETURN(hipFree(d_y));
	//GPUNet::add_gpu_mem(-(n_hidden+1)*n_output*sizeof(float));
}

void GPUNet::feed_forward_v2_2() {
	int threads = 128;

	float* d_sums_l1, *d_y;
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_sums_l1, (n_input+1)*n_hidden*sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_y, (n_input+1)*n_hidden*sizeof(float)));

	feed_forward_layer_v2_2<<<(n_input+threads-1)/threads, threads>>>(n_input, n_hidden, d_input, d_hidden, d_ih_weights, d_sums_l1);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	d_sums_l1 = reduce(n_hidden, (n_input+1)*n_hidden, d_sums_l1, d_y);

	compute_activation_v2<<<(n_hidden+threads-1)/n_hidden, threads>>>(d_hidden, d_sums_l1, n_hidden, n_input+1);

	// must finish previous layer before computing next
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	CUDA_CHECK_RETURN(hipFree(d_sums_l1));
	//GPUNet::add_gpu_mem(-(n_input+1)*n_hidden*sizeof(float));

	float *d_sums_l2;
	CUDA_CHECK_RETURN(hipMalloc(&d_sums_l2, n_hidden*n_output*sizeof(float)));
	CUDA_CHECK_RETURN(hipMemset(d_y, 0, n_input*n_hidden*sizeof(float)));
	//GPUNet::add_gpu_mem((n_hidden+1)*n_output*sizeof(float));

	feed_forward_layer_v2_2<<<(n_hidden+threads-1)/threads, threads>>>(n_hidden, n_output, d_hidden, d_output, d_ho_weights, d_sums_l2);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	d_sums_l2 = reduce(n_output, (n_hidden+1)*n_output, d_sums_l2, d_y);
	compute_activation_v2<<<(n_output+threads-1)/n_output, threads>>>(d_output, d_sums_l2, n_output, n_hidden+1);

	//sync before measuring time
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	CUDA_CHECK_RETURN(hipFree(d_sums_l2));
	CUDA_CHECK_RETURN(hipFree(d_y));
	//GPUNet::add_gpu_mem(-(n_hidden+1)*n_output*sizeof(float));
}

bool GPUNet::validate_output(float* desired_output) {
	//copy output back to host
	CUDA_CHECK_RETURN(hipMemcpy(h_output, d_output, n_output*sizeof(float), hipMemcpyDeviceToHost));

	for (int i = 0; i < n_output; ++i) {
		//std::cout << "actual = " << desired_output[i] << ", calc = " << h_output[i] << std::endl;
		if (abs(desired_output[i] - h_output[i]) > .005)
			return false;
	}
	return true;
}

bool GPUNet::validate_weights(float *desired_ih_weights, float *desired_ho_weights) {
	//copy inp hid weights to host

	CUDA_CHECK_RETURN(hipMemcpy(h_ih_weights, d_ih_weights, (n_input+1)*n_hidden*sizeof(float), hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipMemcpy(h_ho_weights, d_ho_weights, (n_hidden+1)*n_output*sizeof(float), hipMemcpyDeviceToHost));

	for (int i = 0; i < (n_input+1)*n_hidden; ++i) {
		if (abs(desired_ih_weights[i] - h_ih_weights[i]) > .005)
			return false;
	}

	for (int i = 0; i < (n_hidden+1)*n_output; ++i) {
		if (abs(desired_ho_weights[i] - h_ho_weights[i]) > .005)
			return false;
	}

	return true;
}


void GPUNet::test_feed_forward(Net &net, NetData &d) {
	clock_t start, finish;

	std::cout << "feed forward CPU" << std::endl;
	start = clock();
	net.feed_forward(d.get_training_dataset()->training_set[0]->input);
	finish = clock();
	std::cout << "feed forward CPU time: " << ((float)(finish-start)) / CLOCKS_PER_SEC << "s\n\n";
	//net.print_network();

	std::cout << "Testing method 1" << std::endl;
	feed_forward_v1();
	std::cout << "Validates: " << validate_output(net.outputNeurons) << "\n";
	CUDA_CHECK_RETURN(hipMemset(d_output, 0, n_output*sizeof(float)));

	//print_net();

	std::cout << "Testing method 1.2" << std::endl;
	FeatureVector **dv;
	GPUNet::copy_to_device_host_array_ptrs_biased(d.get_training_dataset()->training_set, &dv);
	feed_forward_v1_2(dv[0]->input);
	std::cout << "Validates: " << validate_output(net.outputNeurons) << "\n";
	//net.print_network();
	//print_net();
	CUDA_CHECK_RETURN(hipMemset(d_output, 0, n_output*sizeof(float)));

	/*std::cout << "Testing method 2" << std::endl;
	feed_forward_v2();
	std::cout << "Validates: " << validates(net.outputNeurons) << "\n";
	CUDA_CHECK_RETURN(hipMemset(d_output, 0, n_output*sizeof(float)));

	std::cout << "Testing method 2.2" << std::endl;
	feed_forward_v2_2();
	std::cout << "Validates: " << validates(net.outputNeurons) << "\n";
	CUDA_CHECK_RETURN(hipMemset(d_output, 0, n_output*sizeof(float)));*/
}

void GPUNet::test_backprop(Net &net, NetData &d) {
	NetTrainer nt(&net);
	//std::cout << "CPU net 0" << std::endl;
	//net.print_network();

	net.feed_forward(d.get_training_dataset()->training_set[0]->input);
	//std::cout << "CPU net 1" << std::endl;
	//net.print_network();

	nt.backprop(d.get_training_dataset()->training_set[0]->target);
	//std::cout << "CPU net 2" << std::endl;
	//net.print_network();

	std::cout << "Testing backprop_v2" << std::endl;
	FeatureVector **dv;
	GPUNet::copy_to_device_host_array_ptrs_biased(d.get_training_dataset()->training_set, &dv);

	//std::cout << std::endl << "GPU net 0" << std::endl;
	//print_net();
	//std::cout << std::endl;

	feed_forward_v1_2(dv[0]->input);
	//std::cout << "GPU net 1" << std::endl;
	//print_net();
	//std::cout << std::endl;

	//std::cout << "GPU net 2" << std::endl;
	backprop_v2(dv[0]->input, dv[0]->target);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	//print_net();
	//std::cout << std::endl;
	std::cout << "Validates: " << validate_weights(net.wInputHidden, net.wHiddenOutput) << std::endl;

//	net.feed_forward(d.get_training_dataset()->training_set[1]->input);
//	nt.backprop(d.get_training_dataset()->training_set[1]->target);
//	nt.update_weights();
//	feed_forward_v1_2(dv[1]->input);
//	backprop_v2(dv[1]->input, dv[1]->target);
//
//
//	std::cout << "Validates: " << validate_weights(net.wInputHidden, net.wHiddenOutput) << std::endl;
}

void GPUNet::run_parallel(Net &net, NetData &d) {
	std::cout << "Running in parallel" <<std::endl;

	FeatureVector **dv;
	GPUNet::copy_to_device_host_array_ptrs_biased(d.get_training_dataset()->training_set, &dv);

	NetTrainer nt(&net);

	int e = 0;
	std::string r = "";
	while (true) {
		std::cout << "Epoch " << e++ << std::endl;
		for (int i = 0; i < d.get_training_dataset()->training_set.size(); ++i) {
			net.feed_forward(d.get_training_dataset()->training_set[i]->input);
			nt.backprop(d.get_training_dataset()->training_set[i]->target);

			feed_forward_v1_2(dv[0]->input);
			backprop_v2(dv[0]->input, dv[0]->target);

			std::cout << "CPU network" << std::endl;
			net.print_network();
			std::cout << "GPU network" << std::endl;
			print_net();
			std::cout << "Validates: " << validate_weights(net.wInputHidden, net.wHiddenOutput) << std::endl;
			std::getline(std::cin, r);
			if (r == "exit") {
				return;
			}
		}
	}
}


void GPUNet::test_reduction() {
	/*
	 * Testing with 4, easy since power of 2
	 */

	std::cout << std::endl << "Reduce array length 4" << std::endl;

	float a[] = {.25, .5, .75, 1};
	float *d_a;
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_a, 4*sizeof(float)));
	CUDA_CHECK_RETURN(hipMemcpy(d_a, &a, 4*sizeof(float), hipMemcpyHostToDevice));
	reduction<<<1, 4>>>(4, 0, 1, d_a);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	CUDA_CHECK_RETURN(hipMemcpy(&a, d_a, 4*sizeof(float), hipMemcpyDeviceToHost));
	for (int i = 0; i < 4; ++i) {
		std::cout << a[i] << " ";
	}
	std::cout << std::endl;

	reduction<<<1, 4>>>(4, 0, 2, d_a);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	CUDA_CHECK_RETURN(hipMemcpy(&a, d_a, 4*sizeof(float), hipMemcpyDeviceToHost));
	for (int i = 0; i < 4; ++i) {
		std::cout << a[i] << " ";
	}
	std::cout << std::endl;
	hipFree(d_a);

	/*
	 * Testing array size 5
	 */

	std::cout << std::endl << "Reduce array length 5" << std::endl;

	float b[] = {.25, .5, .75, 1, 1.25};
	float *d_b;
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_b, 5*sizeof(float)));
	CUDA_CHECK_RETURN(hipMemcpy(d_b, &b, 5*sizeof(float), hipMemcpyHostToDevice));
	reduction<<<1, 5>>>(5, 0, 1, d_b);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	CUDA_CHECK_RETURN(hipMemcpy(&b, d_b, 5*sizeof(float), hipMemcpyDeviceToHost));
	for (int i = 0; i < 5; ++i) {
		std::cout << b[i] << " ";
	}
	std::cout << std::endl;

	reduction<<<1, 5>>>(5, 0, 2, d_b);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	CUDA_CHECK_RETURN(hipMemcpy(&b, d_b, 5*sizeof(float), hipMemcpyDeviceToHost));
	for (int i = 0; i < 5; ++i) {
		std::cout << b[i] << " ";
	}
	std::cout << std::endl;

	reduction<<<1, 5>>>(5, 0, 3, d_b);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	CUDA_CHECK_RETURN(hipMemcpy(&b, d_b, 5*sizeof(float), hipMemcpyDeviceToHost));
	for (int i = 0; i < 5; ++i) {
		std::cout << b[i] << " ";
	}
	std::cout << std::endl;

	hipFree(d_b);


	/*
	 * Testing array size 7
	 */
	std::cout << std::endl << "Reduce array length 7" << std::endl;

	float c[] = {.25, .5, .75, 1, 1.25, 1.5, 1.75};
	float *d_c;
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_c, 7*sizeof(float)));
	CUDA_CHECK_RETURN(hipMemcpy(d_c, &c, 7*sizeof(float), hipMemcpyHostToDevice));

	for (int j = 0; j < ceil(log2(7.0)); ++j) {
		reduction<<<1, 7>>>(7, 0, j+1, d_c);
		CUDA_CHECK_RETURN(hipDeviceSynchronize());
		CUDA_CHECK_RETURN(hipMemcpy(&c, d_c, 7*sizeof(float), hipMemcpyDeviceToHost));
		for (int i = 0; i < 7; ++i) {
			std::cout << c[i] << " ";
		}
		std::cout << std::endl;
	}
	hipFree(d_c);

	/*
	 * testing stacked arrays
	 */
	std::cout << std::endl << "testing stacked arrays 4x4" << std::endl;
	float d[] = {.25, .5, .75, 1, .1, .2, .3, .4, .2, .4, .6, .8, .3, .6, .9, 1.2};
	float *d_d;
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_d, 16*sizeof(float)));
	CUDA_CHECK_RETURN(hipMemcpy(d_d, &d, 16*sizeof(float), hipMemcpyHostToDevice));
	for (int j = 0; j < ceil(log2(4.0)); ++j) {
		reduction<<<4, 4>>>(4, 4, j+1, d_d);
		CUDA_CHECK_RETURN(hipDeviceSynchronize());
		CUDA_CHECK_RETURN(hipMemcpy(&d, d_d, 16*sizeof(float), hipMemcpyDeviceToHost));
		for (int i = 0; i < 16; ++i) {
			std::cout << d[i] << " ";
		}
		std::cout << std::endl;
	}
	hipFree(d_d);


	/*
	 * testing stacked arrays
	 */
	std::cout << std::endl << "testing stacked arrays 4x5" << std::endl;
	float e[] = {.25, .5, .75, 1, 1.25, .1, .2, .3, .4, .5, .2, .4, .6, .8, 1, .3, .6, .9, 1.2, 1.5};
	float *d_e;
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_e, 20*sizeof(float)));
	CUDA_CHECK_RETURN(hipMemcpy(d_e, &e, 20*sizeof(float), hipMemcpyHostToDevice));
	for (int j = 0; j < ceil(log2(5.0)); ++j) {
		reduction<<<4, 5>>>(5, 4, j+1, d_e);
		CUDA_CHECK_RETURN(hipDeviceSynchronize());
		CUDA_CHECK_RETURN(hipMemcpy(&e, d_e, 20*sizeof(float), hipMemcpyDeviceToHost));
		for (int i = 0; i < 20; ++i) {
			std::cout << e[i] << " ";
		}
		std::cout << std::endl;
	}
	hipFree(d_e);
}


size_t GPUNet::current_mem_usage(int dev) {
	return gpu_mem[dev];
}



/*
 * ------------ private ------------
 */


dim3 GPUNet::get_threadsm2l1() {
	dim3 threadsm2l1;
	int s = (int)ceil(sqrt(n_input+1));

	threadsm2l1.x = s;
	threadsm2l1.y = s;

	return threadsm2l1;
}

dim3 GPUNet::get_threadsm2l2() {
	dim3 threadsm2l2;
	int s = (int)ceil(sqrt(n_hidden+1));

	threadsm2l2.x = s;
	threadsm2l2.y = s;

	return threadsm2l2;
}

void GPUNet::add_gpu_mem(int bytes) {
	gpu_mem[get_current_device()] += bytes;
}


int GPUNet::get_current_device() {
	int device;
	hipGetDevice(&device);
	return device;
}

size_t GPUNet::dataset_size(TrainingDataSet *tset) {
	size_t tset_size = 0;
	int fv_size = (n_input + n_output) * sizeof(float);
	tset_size += fv_size * tset->training_set.size();
	tset_size += fv_size * tset->generalization_set.size();
	tset_size += fv_size * tset->validation_set.size();
	return tset_size;
}

size_t GPUNet::total_dev_mem(int dev) {
	hipDeviceProp_t props;
	hipGetDeviceProperties(&props, dev);
	return props.totalGlobalMem - 1611000000; //minus 1.5 gb
}


/*
 * Copies the host vector to a pointer array on the device
 * Cannot index from host
 */
void GPUNet::copy_to_device(thrust::host_vector<FeatureVector*> &hv, FeatureVector ***dv) {

	CUDA_CHECK_RETURN(hipMalloc((void **)&(*dv), hv.size()*sizeof(FeatureVector*)));

	FeatureVector** host_dv_tmp = (FeatureVector**)malloc(hv.size()*sizeof(FeatureVector*));

	for (size_t i = 0; i < hv.size(); ++i) {
		//allocate device memory
		FeatureVector *d_fv;
		CUDA_CHECK_RETURN(hipMalloc((void **)&d_fv, sizeof(FeatureVector)));

		float *d_inp, *d_tar;
		CUDA_CHECK_RETURN(hipMalloc((void **)&d_inp, (n_input)*sizeof(float)));
		CUDA_CHECK_RETURN(hipMalloc((void **)&d_tar, (n_output)*sizeof(float)));

		CUDA_CHECK_RETURN(hipMemcpy(&(d_fv->input), &d_inp, sizeof(float *), hipMemcpyHostToDevice));
		CUDA_CHECK_RETURN(hipMemcpy(d_inp, hv[i]->input, n_input*sizeof(float), hipMemcpyHostToDevice));

		CUDA_CHECK_RETURN(hipMemcpy(&(d_fv->target), &d_tar, sizeof(float *), hipMemcpyHostToDevice));
		CUDA_CHECK_RETURN(hipMemcpy(d_tar, hv[i]->target, n_output*sizeof(float), hipMemcpyHostToDevice));

		for(int a = 0; a < n_input; a++) {
			std::cout << a << ": " << hv[i]->input[a] << std::endl;
		}

		print_floats<<<1,1>>>(n_input, d_inp, d_fv);
		CUDA_CHECK_RETURN(hipDeviceSynchronize());

		std::cout << std::endl;

		host_dv_tmp[i] = d_fv;
	}

	std::cout << "precopy"<<std::endl;
	// Copy to device Memory
	CUDA_CHECK_RETURN(hipMemcpy(*dv, host_dv_tmp, hv.size()*sizeof(FeatureVector*), hipMemcpyHostToDevice));
	std::cout << "postcopy"<<std::endl;

	print_all<<<1,1>>>(n_input, n_output, *dv);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	std::cout << "Copying data 4" << std::endl;
}

/*
 * Copies the host vector to a pointer array on the device
 * Cannot index from host
 * The final bias node is included in the inputs. This is so that when referencing the inputs
 * 	the bias is not lost
 */
void GPUNet::copy_to_device_biased(thrust::host_vector<FeatureVector*> &hv, FeatureVector ***dv) {

	CUDA_CHECK_RETURN(hipMalloc((void **)&(*dv), hv.size()*sizeof(FeatureVector*)));

	FeatureVector** host_dv_tmp = (FeatureVector**)malloc(hv.size()*sizeof(FeatureVector*));

	for (size_t i = 0; i < hv.size(); ++i) {
		//allocate device memory
		FeatureVector *d_fv;
		CUDA_CHECK_RETURN(hipMalloc((void **)&d_fv, sizeof(FeatureVector)));

		float *d_inp, *d_tar;
		//allocate for bias
		CUDA_CHECK_RETURN(hipMalloc((void **)&d_inp, (n_input+1)*sizeof(float)));
		CUDA_CHECK_RETURN(hipMalloc((void **)&d_tar, (n_output)*sizeof(float)));

		CUDA_CHECK_RETURN(hipMemcpy(&(d_fv->input), &d_inp, sizeof(float *), hipMemcpyHostToDevice));
		CUDA_CHECK_RETURN(hipMemcpy(d_inp, hv[i]->input, (n_input)*sizeof(float), hipMemcpyHostToDevice));

		CUDA_CHECK_RETURN(hipMemcpy(&(d_fv->target), &d_tar, sizeof(float *), hipMemcpyHostToDevice));
		CUDA_CHECK_RETURN(hipMemcpy(d_tar, hv[i]->target, n_output*sizeof(float), hipMemcpyHostToDevice));

		//TODO: does setting all in parallel improve speed?
		set_bias<<<1, 1>>>(n_input, d_inp);
		CUDA_CHECK_RETURN(hipDeviceSynchronize());

		for(int a = 0; a < n_input; a++) {
			std::cout << a << ": " << hv[i]->input[a] << std::endl;
		}

		print_floats<<<1,1>>>(n_input+1, d_inp, d_fv);
		CUDA_CHECK_RETURN(hipDeviceSynchronize());

		std::cout << std::endl;

		host_dv_tmp[i] = d_fv;
	}

	std::cout << "precopy"<<std::endl;
    // Copy to device Memory
    CUDA_CHECK_RETURN(hipMemcpy(*dv, host_dv_tmp, hv.size()*sizeof(FeatureVector*), hipMemcpyHostToDevice));
    std::cout << "postcopy"<<std::endl;

	print_all<<<1,1>>>(n_input, n_output, *dv);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	std::cout << "Copying data 4" << std::endl;
}

/*
 * Copies the host vector to a pointer array on the host that holds pointers to head FeatureVector on the device
 */
void GPUNet::copy_to_device_host_array(thrust::host_vector<FeatureVector*> &hv, FeatureVector ***dv) {

	(*dv) = (FeatureVector**)malloc(hv.size()*sizeof(FeatureVector));
	//FeatureVector** host_dv_tmp = (FeatureVector**)malloc(hv.size()*sizeof(FeatureVector*));

	for (size_t i = 0; i < hv.size(); ++i) {
		//allocate device memory
		FeatureVector *d_fv;
		CUDA_CHECK_RETURN(hipMalloc((void **)&d_fv, sizeof(FeatureVector)));

		float *d_inp, *d_tar;
		CUDA_CHECK_RETURN(hipMalloc((void **)&d_inp, (n_input)*sizeof(float)));
		CUDA_CHECK_RETURN(hipMalloc((void **)&d_tar, (n_output)*sizeof(float)));

		CUDA_CHECK_RETURN(hipMemcpy(&(d_fv->input), &d_inp, sizeof(float *), hipMemcpyHostToDevice));
		CUDA_CHECK_RETURN(hipMemcpy(d_inp, hv[i]->input, n_input*sizeof(float), hipMemcpyHostToDevice));

		CUDA_CHECK_RETURN(hipMemcpy(&(d_fv->target), &d_tar, sizeof(float *), hipMemcpyHostToDevice));
		CUDA_CHECK_RETURN(hipMemcpy(d_tar, hv[i]->target, n_output*sizeof(float), hipMemcpyHostToDevice));



		//for(int a = 0; a < n_input; a++) {
		//	std::cout << a << ": " << hv[i]->input[a] << std::endl;
		//}

		//print_floats<<<1,1>>>(n_input, d_inp, d_fv);
		//CUDA_CHECK_RETURN(hipDeviceSynchronize());

		//std::cout << std::endl;

		(*dv)[i] = d_fv;
	}

	//for (int i = 0; i < 4; ++i) {
	//	print_floats2<<<1,1>>>(n_input, (*dv)[i]);
	//	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	//}

}

/*
 * Copies the host vector to a pointer array on the host that holds pointers to FeatureVector on the device with bias node
 */
void GPUNet::copy_to_device_host_array_biased(thrust::host_vector<FeatureVector*> &hv, FeatureVector ***dv) {

	(*dv) = (FeatureVector**)malloc(hv.size()*sizeof(FeatureVector));
	//FeatureVector** host_dv_tmp = (FeatureVector**)malloc(hv.size()*sizeof(FeatureVector*));

	for (size_t i = 0; i < hv.size(); ++i) {
		//allocate device memory
		FeatureVector *d_fv;
		CUDA_CHECK_RETURN(hipMalloc((void **)&d_fv, sizeof(FeatureVector)));

		float *d_inp, *d_tar;
		CUDA_CHECK_RETURN(hipMalloc((void **)&d_inp, (n_input+1)*sizeof(float)));
		CUDA_CHECK_RETURN(hipMalloc((void **)&d_tar, (n_output)*sizeof(float)));

		CUDA_CHECK_RETURN(hipMemcpy(&(d_fv->input), &d_inp, sizeof(float *), hipMemcpyHostToDevice));
		CUDA_CHECK_RETURN(hipMemcpy(d_inp, hv[i]->input, n_input*sizeof(float), hipMemcpyHostToDevice));

		CUDA_CHECK_RETURN(hipMemcpy(&(d_fv->target), &d_tar, sizeof(float *), hipMemcpyHostToDevice));
		CUDA_CHECK_RETURN(hipMemcpy(d_tar, hv[i]->target, n_output*sizeof(float), hipMemcpyHostToDevice));


		//TODO: does setting all in parallel improve speed?
		set_bias<<<1, 1>>>(n_input, d_inp);
		CUDA_CHECK_RETURN(hipDeviceSynchronize());

		for(int a = 0; a < n_input; a++) {
			std::cout << a << ": " << hv[i]->input[a] << std::endl;
		}

		print_floats<<<1,1>>>(n_input+1, d_inp, d_fv);
		CUDA_CHECK_RETURN(hipDeviceSynchronize());

		std::cout << std::endl;

		//for(int a = 0; a < n_input; a++) {
		//	std::cout << a << ": " << hv[i]->input[a] << std::endl;
		//}

		//print_floats<<<1,1>>>(n_input, d_inp, d_fv);
		//CUDA_CHECK_RETURN(hipDeviceSynchronize());

		//std::cout << std::endl;

		(*dv)[i] = d_fv;
	}

	//for (int i = 0; i < 4; ++i) {
	//	print_floats2<<<1,1>>>(n_input, (*dv)[i]);
	//	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	//}
}


/*
 * Copies the host vector to a pointer array on the host that holds pointers to FeatureVector on the device with bias node
 */
void GPUNet::copy_to_device_host_array_ptrs_biased(thrust::host_vector<FeatureVector*> &hv, FeatureVector ***dv) {

	*dv = (FeatureVector**)malloc(hv.size()*sizeof(FeatureVector*));
	//FeatureVector** host_dv_tmp = (FeatureVector**)malloc(hv.size()*sizeof(FeatureVector*));

	for (size_t i = 0; i < hv.size(); ++i) {
		//allocate device memory
		FeatureVector *d_fv = (FeatureVector*)malloc(sizeof(FeatureVector*));

		float *d_inp, *d_tar;
		CUDA_CHECK_RETURN(hipMalloc((void **)&d_inp, (n_input+1)*sizeof(float)));
		CUDA_CHECK_RETURN(hipMalloc((void **)&d_tar, (n_output)*sizeof(float)));

		CUDA_CHECK_RETURN(hipMemcpy(d_inp, hv[i]->input, n_input*sizeof(float), hipMemcpyHostToDevice));
		CUDA_CHECK_RETURN(hipMemcpy(d_tar, hv[i]->target, n_output*sizeof(float), hipMemcpyHostToDevice));

		d_fv->input = d_inp;
		d_fv->target = d_tar;

		//TODO: does setting all in parallel improve speed?
		set_bias<<<1, 1>>>(n_input, d_inp);
		CUDA_CHECK_RETURN(hipDeviceSynchronize());

		(*dv)[i] = d_fv;
	}

}

/**
 * Copy from pattern p_start to p_end to device
 * only allocate memory if \p allocate is true
 */
void GPUNet::copy_to_device_host_array_ptrs_biased_section(thrust::host_vector<FeatureVector*> &hv, FeatureVector ***dv,
		int p_start, int p_end, bool allocate) {

	if (allocate) { // if the first epoch and the first section
		*dv = (FeatureVector**)malloc(hv.size()*sizeof(FeatureVector*));
	}

	for (int i = p_start, p = 0; i < p_end; ++i, ++p) {
		if (allocate) {
			//allocate device memory
			FeatureVector *d_fv = (FeatureVector*)malloc(sizeof(FeatureVector*));

			float *d_inp, *d_tar;
			CUDA_CHECK_RETURN(hipMalloc((void **)&d_inp, (n_input+1)*sizeof(float)));
			CUDA_CHECK_RETURN(hipMalloc((void **)&d_tar, (n_output)*sizeof(float)));

			CUDA_CHECK_RETURN(hipMemcpy(d_inp, hv[i]->input, n_input*sizeof(float), hipMemcpyHostToDevice));
			CUDA_CHECK_RETURN(hipMemcpy(d_tar, hv[i]->target, n_output*sizeof(float), hipMemcpyHostToDevice));

			d_fv->input = d_inp;
			d_fv->target = d_tar;

			//TODO: does setting all in parallel improve speed?
			set_bias<<<1, 1>>>(n_input, d_inp);
			CUDA_CHECK_RETURN(hipDeviceSynchronize());
			(*dv)[p] = d_fv;
		} else {
			CUDA_CHECK_RETURN(hipMemcpy((*dv)[p]->input, hv[i]->input, n_input*sizeof(float), hipMemcpyHostToDevice));
			CUDA_CHECK_RETURN(hipMemcpy((*dv)[p]->target, hv[i]->target, n_output*sizeof(float), hipMemcpyHostToDevice));
		}

	}
}
