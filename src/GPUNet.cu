#include "hip/hip_runtime.h"
/*
 * GPUNet.cpp
 *
 *  Created on: Jan 5, 2014
 *      Author: trevor
 */

#include "GPUNet.h"
#include "NetTrainer.h"
#include "NetIO.h"
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <thrust/device_vector.h>
#include "hiprand/hiprand_kernel.h"

/*
 * ------------ CUDA ------------
 */


/**
 * Get a random number within a given float range
 * \param min float
 * \param max float
 * \param i int
 * \param *global hiprandState
 */
__device__ float get_random_range(float min, float max, int i, hiprandState *global) {
	hiprandState local = global[i];
	float r = hiprand_uniform(&local);
	global[i] = local;
	return min + r * (max - min);
}

__device__ float get_ih_weight(float* ih_weights, int n_hidden, int i, int h) {
	return ih_weights[n_hidden*i + h];
}

__device__ float get_ho_weight(float* ho_weights, int n_output, int h, int o) {
	return ho_weights[n_output*h + o];
}

/**
 * Compute the sigmoid value of a given float
 * \param x the value to compute the sigmoid of
 */
__device__ inline float sigmoid(float x) {
	return 1.0 / (1.0 + exp(-x));
}


/**
 * Compute the output gradient given specific output and target values
 * \param output float
 * \param target float
 */
__device__ float calc_output_gradient(float output, float target) {
	return output * (1 - output) * (target - output);
}


/**
 * Clamp the output to 0 or 1 if within .1
 *\param f the value to clamp
 */
__device__ int clamp(float f) {
	if (f < .1) {
		return 0;
	} else if (f > .9) {
		return 1;
	} else {
		return -1;
	}
}

/*
 *
 * ------------- Initialization kernels ---------------
 *
 */


/**
 * Initialize random seeds in CUDA
 */
__global__ void curand_setup(hiprandState *state) {
	unsigned int seed = (unsigned int)clock64();
	int id = threadIdx.x;
	hiprand_init(seed, id, 0, &state[id]);
}

__global__ void curand_setup_v2(int n, hiprandState *state) {
	unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < n) {
		unsigned int seed = (unsigned int)clock64();
		hiprand_init(seed, id, 0, &state[id]);
	}
}

/**
 * initialize nodes to 0 or 1 if bias
 * generic
 */
__global__ void init_nodes_layer_v2(int n, float *nodes) {
	unsigned int i = blockIdx.x * blockDim.x+threadIdx.x;
	if (i < n) {
		if (i == n-1)
			nodes[i] = 1;
		else
			nodes[i] = 0;
	}
}

/**
 * set all output nodes to 0
 */

__global__ void init_nodes_output_v2(int n, float *output) {
	unsigned int i = blockIdx.x * blockDim.x+threadIdx.x;
	if (i < n) {
		output[i] = 0;
	}
}


__global__ void init_weights_v2(int n1, int n2, float *weights, hiprandState *state) {
	unsigned int i = blockIdx.x * blockDim.x+threadIdx.x;
	// r is the range for random values
	if (i < (n1+1)*n2) {
		float r = 1.0 / sqrt((float)blockDim.x-1);
		int node_l1 = i % (n1+1);
		int node_l2 = i % n2;
		weights[n2*node_l1 + node_l2] = get_random_range(-r, r, n2*node_l1 + node_l2, state);
	}
}


__global__ void init_deltas_v2(unsigned int n1, unsigned int n2, float *deltas) {
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < (n1+1)*n2) {
		deltas[i] = 0;
	}
}



/* --------------- Referencing and simple set function ---------------
 * set bias
 *
 */

//used when copying patterns to device
__global__ void set_bias(int n_input, float *d_inp) {
	d_inp[n_input] = 1;
}


/*
 * -------------- Error calculation ---------------
 * output_correct
 * mse_sum
 *
 */

__device__ int d_num_correct = 0;
__device__ float d_acc = 0;
__device__ float d_mse_sum = 0;
__device__ float d_mse = 0; //current mse

__global__ void output_correct_v2(float *output, float *target, int n_output) {
	for (int i = 0; i < n_output; ++i) {
		if (clamp(output[i]) != clamp(target[i])) {
			return;
		}
	}
	++d_num_correct;
}

__global__ void calc_acc(int n_patterns) {
	d_acc = ((float)d_num_correct/n_patterns * 100);
	d_num_correct = 0;
}

/**
 * single threaded
 */
__global__ void mse_sum_v2(float *output, float *target, int n_output) {
	float sum = 0;
	for (int i = 0; i < n_output; ++i) {
		sum += pow(output[i] - target[i], 2);
	}
	d_mse_sum += sum;
}

/**
 * single threaded
 */
__global__ void calc_mse(int n_output, int n_patterns) {
	d_mse = d_mse_sum / (n_output * n_patterns);
	d_mse_sum = 0;
}



/*
 * ---- feed forward kernels -----------
 *
 * method 1 calculates each node in the next layer with a single thread computing for each output node
 * method 2 has a thread for each term in the linear combination to compute the output
 *     then the activation is computed after syncing threads.
 */

/*
 * to measure bandwidth:
 * (bytes read + bytes writen) / (time secs * 10^9) = gb
 *
 * bytes read = 4* ((n_layer1+1)*2),
 * bytes written = 4* (n_layer2)
 * total/thread = 4*((n_layer1+1)*2 + n_layer2)
 * threads l1 -> l2 = n_hidden
 * threads l2 -> l3 = n_output
 *
 * total_l1->l2 = n_hidden*4*((n_layer1+1)*2 + n_layer2)
 * total_l2->l3 = n_output*4*((n_layer2+1)*2 + n_layer3)
 *
 * total = total_l1->l2 + total_l2->l3;
 */
__global__ void feed_forward_layer_v1(int n_layer1, int n_layer2, float* layer1, float* layer2, float* weights) {
	int n = threadIdx.x; // node to compute;

	float r = 0;
	for (int i = 0; i <= n_layer1; ++i) { //include bias
		r += layer1[i] * weights[n_layer2*i + n];
	}
	layer2[n] = sigmoid(r);
}

/*
 * Generic version, called with pow of 2 threads
 */
__global__ void feed_forward_layer_v1_2(int n_layer1, int n_layer2, float* layer1, float* layer2, float* weights) {
	unsigned int n = blockIdx.x * blockDim.x+threadIdx.x; // node to compute;
	if (n < n_layer2) {
		float r = 0;
		for (int i = 0; i <= n_layer1; ++i) { //include bias
			r += layer1[i] * weights[n_layer2*i + n];
		}
		layer2[n] = sigmoid(r);
	}
}


__global__ void feed_forward_layer_v2_2(unsigned int pow2, int n_layer1, int n_layer2, float* layer1, float* layer2, float* weights, float* sums) {
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x; // input node
	//printf("x = %d\n", x);
	if (x < (n_layer1+1)*n_layer2) {
		//printf("x = %d\n", x);
		int i = x % (n_layer1+1);
		int j = i % n_layer2;
		int p = j*pow2 + i;
		sums[p] = layer1[i] * weights[n_layer2*i + j];
	}
}

/*
 * generic version
 */
__global__ void compute_activation_v2(float* nodes, float *sums, int n_layer, int stride) {
	unsigned int i = blockIdx.x * blockDim.x+threadIdx.x; // input node

	if (i < n_layer)
		nodes[i] = sigmoid(sums[i*stride]);
}

__global__ void clamp_outputs(float *output, int n) {
	unsigned int i = blockIdx.x * blockDim.x+threadIdx.x;
	if (i < n) {
		output[i] = clamp(output[i]);
	}
}

/*
 * Copied form NVIDIA presentation
 * http://developer.download.nvidia.com/compute/cuda/1.1-Beta/x86_website/projects/reduction/doc/reduction.pdf
 */
template <unsigned int blockSize>
__global__ void reduce_kernel(float *g_idata, float *g_odata, unsigned int n, int offset) {
	g_idata = &(g_idata[n*offset]);

	__syncthreads();
	extern __shared__ float sdata[];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockSize*2) + tid;
	unsigned int gridSize = blockSize*2*gridDim.x;
	sdata[tid] = 0;
	while (i < n) { sdata[tid] += g_idata[i] + g_idata[i+blockSize]; i += gridSize; }
	__syncthreads();
	if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
	if (blockSize >= 256) {if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
	if (blockSize >= 128) {if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
	if (tid < 32) { if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
		if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
		if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
		if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
		if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
		if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
	}
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}


/*
 *
 *
 * ------------ backprop kernels ---------
 *
 *
 */


/*
 * called generically, pow of 2 threads
 */
__global__ void output_error_gradients_v2(float* output, float* target, float* output_err_gradients, int no) {
	unsigned int i = blockIdx.x * blockDim.x+threadIdx.x;

	if (i < no) {
		output_err_gradients[i] = calc_output_gradient(output[i], target[i]);
		//printf("out_err_grad[%d] = %f, output = %f, target = %f\n", i, output_err_gradients[i], output[i], target[i]);
	}
}


/*
 * called generically with power of 2 threads
 */
__global__ void update_hidden_output_deltas_v2(int nh, int no, float l_rate, float momentum,
		float* hidden, float* output_err_gradients, float* delta_ho) {

	unsigned int x = blockIdx.x * blockDim.x+threadIdx.x;

	if (x < (nh+1)*no) { // if in range
		//NOTE: this was my bug, had (x % nh) not (x % (nh+1))
		int j = x % (nh+1); //input node
		int k = x % no; //hidden node

		delta_ho[no*j + k] = l_rate * hidden[j] * output_err_gradients[k] + momentum * delta_ho[no*j + k];
		//printf("delta_ho(%d, %d) = %f, l_rate = %f, hidden[%d] = %f, out_err_gradients[%d] = %f, momentum = %f\n",
		//			j, k, delta_ho[no*j+k], l_rate, j, hidden[j], k, output_err_gradients[k], momentum);
	}
}


__device__ float calc_hidden_gradient(int j, int no, float* hidden, float* d_ho_weights, float* output_err_gradients) {
	//get sum of hidden->output weights * output error gradients
	float s = 0;
	for (int k = 0; k < no; ++k)
		s += d_ho_weights[j*no + k] * output_err_gradients[k];

	//return error gradient
	return hidden[j] * (1 - hidden[j]) * s;
}

/*
 * called generically, pow of 2 threads
 */
__global__ void hidden_error_gradients_v2(int nh, int no, float* hidden, float* d_ho_weights, float* hidden_err_gradients, float* output_err_gradients) {
	unsigned int j = blockIdx.x * blockDim.x+threadIdx.x;

	if (j < nh) { //NOTE: another bug, had (j < (nh+1)*no), only nh nodes need calculated
		hidden_err_gradients[j] = calc_hidden_gradient(j, no, hidden, d_ho_weights, output_err_gradients);
		//printf("hidden_err_grad[%d] = %f\n", j, hidden_err_gradients[j]);
	}
}

/*
 * called with any number of blocks / threads
 * normally, 128 or other power of 2
 */
//TODO: perhaps there is a way to store the hidden_err_gradient[j] in shared memory
__global__ void update_input_hidden_deltas_v2(int ni, int nh, float l_rate, float momentum,
		float* input, float* hidden_err_gradients, float* delta_ih) {
	unsigned int x = blockIdx.x * blockDim.x+threadIdx.x;

	if (x < (ni+1)*nh) {
		int i = x % (ni+1); //input node, NOTE: same bug as before
		int j = x % nh; //hidden node

		delta_ih[nh*i + j] = l_rate * input[i] * hidden_err_gradients[j] + momentum * delta_ih[nh*i + j];

		//printf("delta_ih(%d, %d) = %f, l_rate = %f, input[%d] = %f, hidden_err_gradients[%d] = %f, momentum = %f\n",
		//			i, j, delta_ih[nh*i + j], l_rate, i, input[i], j, hidden_err_gradients[j], momentum);
	}
}



/*
 * weight update
 */

/*
 * called generically with power of 2 threads
 */
__global__ void update_weights_v2(int n1, int n2, float *d_weights, float *deltas) {
	unsigned int x = blockIdx.x * blockDim.x+threadIdx.x;

	if (x < (n1+1)*n2) {
		int i = x % (n1+1); //layer 1 node, NOTE: same bug
		int j = x % n2; //layer 2 node

		d_weights[i*n2 + j] += deltas[i*n2 + j];
		//printf("d_weights(%d, %d) = %f, deltas(%d, %d) = %f\n", i, j, d_weights[n2*i+j], i, j, deltas[n2*i + j]);
	}
}

__global__ void print_gpu_net(int n_input, int n_hidden, int n_output,
		float *input, float *hidden, float *output, float *ih_weights, float *ho_weights) {
	for (int i = 0; i <= n_input; ++i) {
		printf("input %d: %f, ", i, input[i]);
	}
	printf("\n");
	for (int i = 0; i <= n_input; ++i) {
		for (int j = 0; j < n_hidden; ++j) {
			printf("ih weight (%d, %d): %f, ", i, j, ih_weights[n_hidden*i + j]);
		}
	}
	printf("\n");
	for (int i = 0; i <= n_hidden; ++i) {
		printf("hidden %d: %f, ", i, hidden[i]);
	}
	printf("\n");
	for (int i = 0; i <= n_hidden; ++i) {
		for (int j = 0; j < n_output; ++j) {
			printf("ho weight (%d, %d): %f, ", i, j, ho_weights[n_output*i + j]);
		}
	}
	printf("\n");
	for (int i = 0; i < n_output; ++i) {
		printf("output %d: %f, ", i, output[i]);
	}
	printf("\n");
}

/*
 *
 * --------- Debugging ------------
 *
 */

__global__ void print_target(int n_output, float *target) {
	for (int i = 0; i < n_output; ++i) {
		printf("target[%d] = %f\n", i, target[i]);
	}
}

__global__ void print_input(int n_input, float *input) {
	for (int i = 0; i < n_input+1; i++) {
		printf("input[%d] = %f\n", i, input[i]);
	}
}

/*
 * ---------- Constructors -------------
 */

GPUNet::GPUNet() {
	n_input = 0;
	GPUNet::init_structure(0, 0, GPUNet::STANDARD);
	GPUNet::init_vars();
}

GPUNet::GPUNet(int ni, int no, GPUNet::NetworkStructure net_type) {
	n_input = 0;
	GPUNet::init_structure(ni, no, net_type);
	GPUNet::init_vars();
}

GPUNet::GPUNet(std::string net_file) {
	std::cout << "Initializing from net file: " << net_file << "." << std::endl;
	init_vars();
	read_net(net_file);
}

GPUNet::~GPUNet() {
	hipFree(d_input);
	hipFree(d_hidden);
	hipFree(d_output);
	hipFree(d_target);
	hipFree(d_ih_weights);
	hipFree(d_ho_weights);
	hipFree(d_ih_deltas);
	hipFree(d_ho_deltas);
	hipFree(d_hid_err_gradients);
	hipFree(d_out_err_gradients);

	CUDA_CHECK_RETURN(hipStreamDestroy(err_calc_stream));
	CUDA_CHECK_RETURN(hipStreamDestroy(weight_update_stream));
	CUDA_CHECK_RETURN(hipEventDestroy(event1));

	delete[] h_output;
	delete[] gpu_mem;
}

/*
 * -------------- public ---------------
 */


void GPUNet::init_structure(unsigned int ni, unsigned int no, GPUNet::NetworkStructure net_type) {
	if (n_input != 0) { // constructor initializing nodes has been called, error out
		std::cerr << "Network has already been initialized" << std::endl;
	} else if (ni != 0) { // if not empty constructor
		n_input = ni;
		n_output = no;
		GPUNet::net_type = net_type;
		if (net_type == GPUNet::STANDARD) {
			n_hidden = ceil(2.0/3.0*ni);
		} else if (net_type == GPU_ARCH_OPT) {
			n_hidden = 128 / (2.0/3.0*ni) * 128;
		} else {
			std::cerr << "Invalid network type: " << net_type << std::endl;
			exit(1);
		}
	}
}

void GPUNet::init_vars() {
	max_epochs = GPU_MAX_EPOCHS;
	l_rate = GPU_LEARNING_RATE;
	momentum = GPU_MOMENTUM;
	desired_acc = GPU_DESIRED_ACCURACY;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&n_gpus));

	epoch = 0;
	trainingSetAccuracy = 0;
	validationSetAccuracy = 0;
	generalizationSetAccuracy = 0;
	trainingSetMSE = 0;
	validationSetMSE = 0;
	generalizationSetMSE = 0;

	start = 0;
	finish = 0;

	/*
	 * device
	 */
	d_input = NULL;
	d_hidden = NULL;
	d_output = NULL;
	d_target = NULL;

	d_ih_weights = NULL;
	d_ho_weights = NULL;

	d_ih_deltas = NULL;
	d_ho_deltas = NULL;

	d_hid_err_gradients = NULL;
	d_out_err_gradients = NULL;

	CUDA_CHECK_RETURN(hipStreamCreate(&err_calc_stream));
	CUDA_CHECK_RETURN(hipStreamCreate(&weight_update_stream));
	CUDA_CHECK_RETURN(hipEventCreate(&event1));

	/*
	 * host validation
	 */
	h_output = new float[n_output];
	h_ih_weights = new float[(n_input+1)*n_hidden];
	h_ho_weights = new float[(n_hidden+1)*n_output];

	//init gpu mem to 0 for each gpu
	gpu_mem = new size_t[n_gpus];
	memset(gpu_mem, 0, n_gpus*sizeof(size_t));
	//for (int i = 0; i < n_gpus; ++i) {
	//	gpu_mem[i] = 0;
	//}
}


/*
 * allocate memory on device for
 * input, hidden, output, target
 * ih_weights, ho_weights
 * ih_deltas, ho_deltas
 * hid_err_gradients
 * out_err_gradients
 */

void GPUNet::alloc_dev_mem() {
	//nodes
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_input, (n_input+1)*sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_hidden, (n_hidden+1)*sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_output, (n_output)*sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_target, (n_output)*sizeof(float)));
	add_gpu_mem((n_input+n_hidden+(2*n_output)+2)*sizeof(float));

	//weights
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_ih_weights, ((n_input+1)*n_hidden)*sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_ho_weights, ((n_hidden+1)*n_output)*sizeof(float)));
	add_gpu_mem(((n_input+1)*n_hidden + (n_hidden+1)*n_output)*sizeof(float));

	//create delta arrays, include bias
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_ih_deltas, ((n_input+1)*n_hidden)*sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_ho_deltas, ((n_hidden+1)*n_output)*sizeof(float)));
	add_gpu_mem(((n_input+1)*n_hidden + (n_hidden+1)*n_output)*sizeof(float));

	//error gradients
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_hid_err_gradients, (n_hidden+1)*sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_out_err_gradients, (n_output+1)*sizeof(float)));
	add_gpu_mem((n_hidden + n_output + 2)*sizeof(float));

	std::cout << "Memory allocated on device" << std::endl;
}

/*
 * Note: assumes sizes of networks are the same
 * This is for testing purposes so that
 * I can have identical networks.
 */
void GPUNet::init_from_net(Net &net, NetData &d) {
	int threads = 128;

	//copy first pattern to input neurons so it is copied to device, instead of zeros
	for (int i = 0; i < net.n_input; ++i) {
		net.inputNeurons[i] = d.get_training_dataset()->training_set[0]->input[i];
	}

	// so hidden and output initialized to 0
	CUDA_CHECK_RETURN(hipMemcpy(d_input, net.inputNeurons, (net.n_input)*sizeof(float), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_hidden, net.hiddenNeurons, (net.n_hidden)*sizeof(float), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_output, net.outputNeurons, (net.n_output)*sizeof(float), hipMemcpyHostToDevice));

	set_bias<<<1,1>>>(n_input, d_input);
	set_bias<<<1,1>>>(n_hidden, d_hidden);

	CUDA_CHECK_RETURN(hipMemcpy(d_ih_weights, net.wInputHidden, (net.n_input+1)*(net.n_hidden)*sizeof(float), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_ho_weights, net.wHiddenOutput, (net.n_hidden+1)*(net.n_output)*sizeof(float), hipMemcpyHostToDevice));

	init_deltas_v2<<<((n_input+1)*n_hidden+threads-1)/threads, threads>>>(n_input+1, n_hidden, d_ih_deltas);
	init_deltas_v2<<<((n_hidden+1)*n_output+threads-1)/threads, threads>>>(n_hidden+1, n_output, d_ho_deltas);

	std::cout << "Data copied to device" << std::endl << std::endl;
}


void GPUNet::init_net() {
	int threads = 128;

	//init nodes to all 0
	init_nodes_layer_v2<<<(n_input+1+threads-1)/threads, threads>>>(n_input+1, d_input);
	init_nodes_layer_v2<<<(n_hidden+1+threads-1)/threads, threads>>>(n_hidden+1, d_hidden);
	init_nodes_output_v2<<<(n_output+threads-1)/threads, threads>>>(n_output, d_output);

	//init weights to random vals
	hiprandState *state;
	std::cout << "size of hiprandState: " << sizeof(hiprandState) << std::endl;
	CUDA_CHECK_RETURN(hipMalloc(&state, (n_input+1)*n_hidden*sizeof(hiprandState)));
	curand_setup<<<1, (n_input+1)*n_hidden>>>(state);
	//curand_setup_v2<<<((n_input+1)*n_hidden+threads-1)/threads, threads>>>((n_input+1)*n_hidden, state);

	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	dim3 ih_threads(n_input+1, n_hidden);
	init_weights_v2<<<((n_input+1)*n_hidden+threads-1)/threads, threads>>>(n_input+1, n_hidden, d_ih_weights, state);
	CUDA_CHECK_RETURN(hipFree(state));

	CUDA_CHECK_RETURN(hipMalloc(&state, (n_hidden+1)*n_output*sizeof(hiprandState)));
	curand_setup<<<1, (n_hidden+1)*n_output>>>(state);
	//curand_setup_v2<<<((n_hidden+1)*n_output+threads-1)/threads, threads>>>((n_hidden+1)*n_output, state);

	dim3 ho_threads(n_hidden+1, n_output);
	init_weights_v2<<<((n_hidden+1)*n_output+threads-1)/threads, threads>>>(n_hidden+1, n_output, d_ho_weights, state);
	CUDA_CHECK_RETURN(hipFree(state));

	//init deltas to 0
	init_deltas_v2<<<((n_input+1)*n_hidden+threads-1)/threads, threads>>>(n_input+1, n_hidden, d_ih_deltas);
	init_deltas_v2<<<((n_hidden+1)*n_output+threads-1)/threads, threads>>>(n_hidden+1, n_output, d_ho_deltas);

	std::cout << "net initialized" << std::endl;
}

void GPUNet::set_learning_rate(float lr) {
	l_rate = lr;
}

void GPUNet::set_momentum(float m) {
	momentum = m;
}

void GPUNet::set_training_params(float lr, float m) {
	l_rate = lr;
	momentum = m;
}

void GPUNet::set_max_epochs(int me) {
	max_epochs = me;
}

void GPUNet::set_desired_accuracy(float acc) {
	desired_acc = acc;
}

void GPUNet::set_stopping_conds(int me, float acc) {
	max_epochs = me;
	desired_acc = acc;
}


/*
 * to keep it simple, run in 1 thread
 */
void GPUNet::print_net() {
	print_gpu_net<<<1, 1>>>(n_input, n_hidden, n_output,
			d_input, d_hidden, d_output, d_ih_weights, d_ho_weights);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
}


/*
 * transfer weights back to host
 * write important data (num_epochs, layers, nodes/layer, l_rate, momentum, max_epochs, desired_acc, current mse, current acc)
 *
 */
bool GPUNet::write_net(std::string fname) {
	float *ih_weights = new float[(n_input+1)*(n_hidden)];
	float *ho_weights = new float[(n_hidden+1)*(n_output)];

	CUDA_CHECK_RETURN(hipMemcpy(ih_weights, d_ih_weights, (n_input+1)*(n_hidden)*sizeof(float), hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipMemcpy(ho_weights, d_ho_weights, (n_hidden+1)*(n_output)*sizeof(float), hipMemcpyDeviceToHost));

	NetIO nio;
	nio.epoch = epoch;
	nio.max_epochs = max_epochs;
	nio.net_type = net_type;
	nio.n_input = n_input;
	nio.n_hidden = n_input;
	nio.n_output = n_input;
	nio.l_rate = l_rate;
	nio.momentum = momentum;
	nio.desired_acc = desired_acc;
	nio.trainingSetAccuracy = trainingSetAccuracy;
	nio.generalizationSetAccuracy = generalizationSetAccuracy;
	nio.validationSetAccuracy = validationSetAccuracy;
	nio.trainingSetMSE = trainingSetMSE;
	nio.generalizationSetMSE = generalizationSetMSE;
	nio.validationSetMSE = validationSetMSE;
	nio.ih_weights = ih_weights;
	nio.ho_weights = ho_weights;

	if (!nio.write_net(fname)) {
		std::cout << "Write failed" << std::endl;
		return false;
	}
	delete[] ih_weights;
	delete[] ho_weights;

	return true;
}

bool GPUNet::read_net(std::string fname) {
	NetIO nio;
	if (!nio.read_net(fname)) {
		std::cerr << "Read failed" << std::endl;
		return false;
	}
	epoch = nio.epoch;
	max_epochs = nio.max_epochs;
	net_type = nio.net_type;

	n_input = nio.n_input;
	n_hidden = nio.n_hidden;
	n_output = nio.n_output;
	//now know network size, so allocate
	alloc_dev_mem();

	l_rate = nio.l_rate;
	momentum = nio.momentum;
	desired_acc = nio.desired_acc;
	trainingSetAccuracy = nio.trainingSetAccuracy;
	generalizationSetAccuracy = nio.generalizationSetAccuracy;
	validationSetAccuracy = nio.validationSetAccuracy;
	trainingSetMSE = nio.trainingSetMSE;
	generalizationSetMSE = nio.generalizationSetMSE;
	validationSetMSE = nio.validationSetMSE;

	CUDA_CHECK_RETURN(hipMemcpy(d_ih_weights, nio.ih_weights, (n_input+1)*n_hidden*sizeof(float), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_ho_weights, nio.ho_weights, (n_hidden+1)*n_output*sizeof(float), hipMemcpyHostToDevice));

	return true;
}

/*
 * run the input through the network
 */
float* GPUNet::evaluate(float* input) {
	//copy to device
	//feed forward
	//copy back output
	int threads = 128;
	float *h_out = new float[n_output];
	CUDA_CHECK_RETURN(hipMemcpy((void*)d_input, (void*)input, n_input*sizeof(float), hipMemcpyHostToDevice));
	feed_forward_v1_2(d_input);
	clamp_outputs<<<(n_output+threads-1)/threads, threads>>>(d_output, n_output);
	CUDA_CHECK_RETURN(hipMemcpy(h_out, d_output, n_output*sizeof(float), hipMemcpyDeviceToHost));
	return h_out;
}

float* GPUNet::batch_evaluate(float** inputs) {
	//copy to device
	//feed forward
	//copy back output
	int threads = 128;
	float *h_out = new float[n_output];
	CUDA_CHECK_RETURN(hipMemcpy((void*)d_input, (void*)inputs, n_input*sizeof(float), hipMemcpyHostToDevice));
	feed_forward_v1_2(d_input);
	clamp_outputs<<<(n_output+threads-1)/threads, threads>>>(d_output, n_output);
	CUDA_CHECK_RETURN(hipMemcpy(h_out, d_output, n_output*sizeof(float), hipMemcpyDeviceToHost));
	return h_out;
}

int GPUNet::get_num_input() {
	return n_input;
}

int GPUNet::get_num_hidden() {
	return n_hidden;
}

int GPUNet::get_num_output() {
	return n_output;
}


void GPUNet::calc_dataset_parameters(TrainingDataSet *tset) {
	std::cout << "Determining data set statistics" << std::endl;
	// calc num patterns copyable
	// num patterns = integer div of available memory / mem for single pattern
	int bytes_per_pattern = sizeof(float)*((n_input+1)+(n_output));
	int cur_dev = get_current_device();
	std::cout << " bytes per pattern = "<<bytes_per_pattern<<std::endl;
	std::cout << " total dev mem = "<< total_dev_mem(cur_dev)<<std::endl;
	std::cout << " current mem usage = "<< current_mem_usage(cur_dev)<<std::endl;
	int available_mem = total_dev_mem(cur_dev) - current_mem_usage(cur_dev);
	std::cout << " available mem = "<<available_mem<<std::endl;
	std::cout << " tset.size = "<<tset->size()<<std::endl;
	n_copyable_patterns = available_mem / bytes_per_pattern;
	if (n_copyable_patterns > tset->size()) {
		n_copyable_patterns = tset->size();
	}
	// calc num sections
	// num_sections = ceil ( n_patterns / n_copyable_patterns)
	n_sections = (tset->size() + n_copyable_patterns - 1) / n_copyable_patterns;
	std::cout << " n_copyable_patterns = "<<n_copyable_patterns<<", n_sections = "<<n_sections<<std::endl<<std::endl;
}



void GPUNet::train_net_sectioned(TrainingDataSet *tset) {
	calc_dataset_parameters(tset);

	std::cout << std::endl << "Neural Network Training Starting: " << std::endl
			<< "----------------------------------------------------" << std::endl
			<< "LR: " << l_rate << ", Momentum: " << momentum << ", Max Epochs: " << max_epochs << std::endl
			<< n_input << " Input Neurons, " << n_hidden << " Hidden Neurons, " << n_output << " Output Neurons" << std::endl
			<< "----------------------------------------------------" << std::endl << std::endl;

	epoch = 0;
	FeatureVector** d_training_set;

	if (n_sections == 1) { // no section copying necessary
		copy_to_device_host_array_ptrs_biased(tset->training_set, &d_training_set);
		while (epoch < max_epochs) {
			run_training_epoch_dev(d_training_set, tset->training_set.size());
			std::cout << "Epoch: " << epoch << std::endl;
			++epoch;
		}
	} else {
		while (epoch < max_epochs) {
			//copy a section and run partial epoch
			for (int i = 0; i < n_sections; ++i) {
				//copy patterns from [n_sections*n_patterns_copyable, (n_sections+1)*n_patterns_copyable)
				int p_start = i * n_copyable_patterns;
				int p_end = p_start + n_copyable_patterns;
				if (p_end > tset->training_set.size()) p_end = tset->training_set.size();
				std::cout << "copying section="<<i<<", pstart="<< p_start << ", pend="<<p_end << std::endl;
				copy_to_device_host_array_ptrs_biased_section(tset->training_set, &d_training_set, p_start, p_end, i == 0 && epoch == 0);
				std::cout << "data copied" << std::endl;
				run_training_epoch_dev(d_training_set, p_end-p_start);
			}

			std::cout << "Epoch: " << epoch << std::endl;
			//once training set is complete increment epoch
			++epoch;
		}
	}

	//out validation accuracy and MSE
	std::cout << std::endl << "Training complete. Elapsed epochs: " << epoch << std::endl;

	CUDA_CHECK_RETURN(hipMemcpyFromSymbol(&trainingSetMSE, HIP_SYMBOL(d_mse), sizeof(float), 0, hipMemcpyDeviceToHost));
	std::cout << "MSE = " << trainingSetMSE << std::endl;
	CUDA_CHECK_RETURN(hipMemcpyFromSymbol(&trainingSetAccuracy, HIP_SYMBOL(d_acc), sizeof(float), 0, hipMemcpyDeviceToHost));
	std::cout << "ACC = " << trainingSetAccuracy << std::endl;

	//free training set
	for (int i = 0; i < tset->training_set.size(); ++i) {
		CUDA_CHECK_RETURN(hipFree(d_training_set[i]->input));
		CUDA_CHECK_RETURN(hipFree(d_training_set[i]->target));
		free(d_training_set[i]);
	}
	free(d_training_set);
}


void GPUNet::run_training_epoch_dev(FeatureVector **feature_vecs, size_t n_features) {
	for (size_t i = 0; i < n_features; ++i) {
		feed_forward_v1_2(feature_vecs[i]->input);
		backprop_v2(feature_vecs[i]->input, feature_vecs[i]->target);
		CUDA_CHECK_RETURN(hipDeviceSynchronize());
	}
	calc_mse<<<1, 1>>>(n_output, n_features);
	calc_acc<<<1, 1>>>(n_features);
	//CUDA_CHECK_RETURN(hipDeviceSynchronize());
	//float mse = 0;
	//CUDA_CHECK_RETURN(hipMemcpyFromSymbol(&mse, HIP_SYMBOL(d_mse), sizeof(float), 0, hipMemcpyDeviceToHost));
	//std::cout << "Current mse = " << mse << std::endl;
}


void GPUNet::backprop_v2(float *d_inp, float *d_tar) {
	int n_threads = 128;

	//maintain mse state
	mse_sum_v2<<<1, 1, 0, err_calc_stream>>>(d_output, d_tar, n_output);
	output_correct_v2<<<1, 1, 0, err_calc_stream>>>(d_output, d_tar, n_output);
	//CUDA_CHECK_RETURN(hipDeviceSynchronize());
	//float mse_sum = 0;
	//CUDA_CHECK_RETURN(hipMemcpyFromSymbol(&mse_sum, HIP_SYMBOL(d_mse_sum), sizeof(float), 0, hipMemcpyDeviceToHost));
	//std::cout << "Current mse_sum = " << mse_sum << std::endl;

	output_error_gradients_v2<<<(n_output+n_threads-1)/n_threads, n_threads>>>(d_output, d_tar, d_out_err_gradients, n_output);
	//CUDA_CHECK_RETURN(hipDeviceSynchronize());

	update_hidden_output_deltas_v2<<<((n_output*(n_hidden+1))+n_threads-1)/n_threads, n_threads>>>(n_hidden, n_output, l_rate, momentum, d_hidden, d_out_err_gradients, d_ho_deltas);
	//CUDA_CHECK_RETURN(hipDeviceSynchronize());

	hidden_error_gradients_v2<<<(n_hidden+n_threads-1)/n_threads, n_threads>>>(n_hidden, n_output, d_hidden, d_ho_weights,
			d_hid_err_gradients, d_out_err_gradients);
	//CUDA_CHECK_RETURN(hipDeviceSynchronize());

	CUDA_CHECK_RETURN(hipEventRecord(event1));
	CUDA_CHECK_RETURN(hipStreamWaitEvent(weight_update_stream, event1, 0));
	update_weights_v2<<<((n_output*(n_hidden+1))+n_threads-1)/n_threads, n_threads, 0, weight_update_stream>>>(n_hidden, n_output, d_ho_weights, d_ho_deltas);

	update_input_hidden_deltas_v2<<<((n_hidden*(n_input+1))+n_threads-1)/n_threads, n_threads>>>(n_input, n_hidden, l_rate, momentum,
			d_inp, d_hid_err_gradients, d_ih_deltas);

	CUDA_CHECK_RETURN(hipEventRecord(event1));
	CUDA_CHECK_RETURN(hipStreamWaitEvent(weight_update_stream, event1, 0));
	update_weights_v2<<<((n_hidden*(n_input+1))+n_threads-1)/n_threads, n_threads, 0, weight_update_stream>>>(n_input, n_hidden, d_ih_weights, d_ih_deltas);
}


void GPUNet::feed_forward_v1_2(float *d_inp) {
	int threads = 128;
	feed_forward_layer_v1_2<<<(n_hidden+threads-1)/threads, threads>>>(n_input, n_hidden, d_inp, d_hidden, d_ih_weights);
	feed_forward_layer_v1_2<<<(n_output+threads-1)/threads, threads>>>(n_hidden, n_output, d_hidden, d_output, d_ho_weights);
}



void GPUNet::feed_forward_v2_2(unsigned int pow2, float *d_inp, float *d_sums) {
	int threads = 128;
	feed_forward_layer_v2_2<<<((n_input+1)*n_hidden+threads-1)/threads, threads>>>(pow2, n_input, n_hidden, d_inp, d_hidden, d_ih_weights, d_sums);


	float *h_sums, *h_tmp;
	h_sums = (float*)malloc(pow2*(n_hidden)*sizeof(float));
	memset(h_sums, 0, pow2*(n_hidden)*sizeof(float));
	h_tmp = (float*)malloc(pow2*sizeof(float));
	memset(h_tmp, 0, pow2*sizeof(float));

	CUDA_CHECK_RETURN(hipMemcpy(h_sums, d_sums, pow2*(n_hidden)*sizeof(float), hipMemcpyDeviceToHost));
	for (int i = 0; i < pow2*(n_hidden); ++i) {
		std::cout << h_sums[i] << " ";
	}
	std::cout << std::endl;

	float *d_tmp;
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_tmp, pow2*sizeof(float)));
	CUDA_CHECK_RETURN(hipMemset(d_tmp, 0, pow2*sizeof(float)));
	//reduce_kernel<128><<<(pow2+threads-1)/threads, threads, threads*sizeof(float)>>>(d_sums, d_tmp, pow2, 0);

	reduce_kernel<128><<<(pow2+threads-1)/threads, threads, threads*sizeof(float)>>>(d_sums, d_tmp, pow2, 1);

	CUDA_CHECK_RETURN(hipMemcpy(h_tmp, d_tmp, pow2*sizeof(float), hipMemcpyDeviceToHost));
	for (int i = 0; i < pow2; ++i) {
		std::cout << h_tmp[i] << " ";
	}
	std::cout << std::endl;

	//compute_activation_v2<<<(n_hidden+threads-1)/n_hidden, threads>>>(d_hidden, d_sums_l1, n_hidden, n_input+1);
}

bool GPUNet::validate_output(float* desired_output) {
	//copy output back to host
	CUDA_CHECK_RETURN(hipMemcpy(h_output, d_output, n_output*sizeof(float), hipMemcpyDeviceToHost));

	for (int i = 0; i < n_output; ++i) {
		//std::cout << "actual = " << desired_output[i] << ", calc = " << h_output[i] << std::endl;
		if (abs(desired_output[i] - h_output[i]) > .005)
			return false;
	}
	return true;
}

bool GPUNet::validate_weights(float *desired_ih_weights, float *desired_ho_weights) {
	//copy inp hid weights to host

	CUDA_CHECK_RETURN(hipMemcpy(h_ih_weights, d_ih_weights, (n_input+1)*n_hidden*sizeof(float), hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipMemcpy(h_ho_weights, d_ho_weights, (n_hidden+1)*n_output*sizeof(float), hipMemcpyDeviceToHost));

	for (int i = 0; i < (n_input+1)*n_hidden; ++i) {
		if (abs(desired_ih_weights[i] - h_ih_weights[i]) > .005)
			return false;
	}

	for (int i = 0; i < (n_hidden+1)*n_output; ++i) {
		if (abs(desired_ho_weights[i] - h_ho_weights[i]) > .005)
			return false;
	}

	return true;
}


void GPUNet::test_feed_forward(Net &net, NetData &d) {
	clock_t start, finish;

	std::cout << "feed forward CPU" << std::endl;
	start = clock();
	net.feed_forward(d.get_training_dataset()->training_set[0]->input);
	finish = clock();
	std::cout << "feed forward CPU time: " << ((float)(finish-start)) / CLOCKS_PER_SEC << "s\n\n";
	//net.print_network();

	std::cout << "Testing method 1.2" << std::endl;
	FeatureVector **dv;
	GPUNet::copy_to_device_host_array_ptrs_biased(d.get_training_dataset()->training_set, &dv);
	feed_forward_v1_2(dv[0]->input);
	std::cout << "Validates: " << validate_output(net.outputNeurons) << "\n";
	//net.print_network();
	//print_net();
	CUDA_CHECK_RETURN(hipMemset(d_output, 0, n_output*sizeof(float)));

	/*std::cout << "Testing method 2" << std::endl;
	feed_forward_v2();
	std::cout << "Validates: " << validates(net.outputNeurons) << "\n";
	CUDA_CHECK_RETURN(hipMemset(d_output, 0, n_output*sizeof(float)));

	std::cout << "Testing method 2.2" << std::endl;
	feed_forward_v2_2();
	std::cout << "Validates: " << validates(net.outputNeurons) << "\n";
	CUDA_CHECK_RETURN(hipMemset(d_output, 0, n_output*sizeof(float)));*/
}

void GPUNet::test_backprop(Net &net, NetData &d) {
	NetTrainer nt(&net);
	//std::cout << "CPU net 0" << std::endl;
	//net.print_network();

	net.feed_forward(d.get_training_dataset()->training_set[0]->input);
	//std::cout << "CPU net 1" << std::endl;
	//net.print_network();

	nt.backprop(d.get_training_dataset()->training_set[0]->target);
	//std::cout << "CPU net 2" << std::endl;
	//net.print_network();

	std::cout << "Testing backprop_v2" << std::endl;
	FeatureVector **dv;
	GPUNet::copy_to_device_host_array_ptrs_biased(d.get_training_dataset()->training_set, &dv);

	//std::cout << std::endl << "GPU net 0" << std::endl;
	//print_net();
	//std::cout << std::endl;

	feed_forward_v1_2(dv[0]->input);
	//std::cout << "GPU net 1" << std::endl;
	//print_net();
	//std::cout << std::endl;

	//std::cout << "GPU net 2" << std::endl;
	backprop_v2(dv[0]->input, dv[0]->target);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	//print_net();
	//std::cout << std::endl;
	std::cout << "Validates: " << validate_weights(net.wInputHidden, net.wHiddenOutput) << std::endl;

//	net.feed_forward(d.get_training_dataset()->training_set[1]->input);
//	nt.backprop(d.get_training_dataset()->training_set[1]->target);
//	nt.update_weights();
//	feed_forward_v1_2(dv[1]->input);
//	backprop_v2(dv[1]->input, dv[1]->target);
//
//
//	std::cout << "Validates: " << validate_weights(net.wInputHidden, net.wHiddenOutput) << std::endl;
}

void GPUNet::run_parallel(Net &net, NetData &d) {
	std::cout << "Running in parallel" <<std::endl;

	FeatureVector **dv;
	GPUNet::copy_to_device_host_array_ptrs_biased(d.get_training_dataset()->training_set, &dv);

	NetTrainer nt(&net);

	int e = 0;
	std::string r = "";
	while (true) {
		std::cout << "Epoch " << e++ << std::endl;
		for (int i = 0; i < d.get_training_dataset()->training_set.size(); ++i) {
			net.feed_forward(d.get_training_dataset()->training_set[i]->input);
			nt.backprop(d.get_training_dataset()->training_set[i]->target);

			feed_forward_v1_2(dv[0]->input);
			backprop_v2(dv[0]->input, dv[0]->target);

			std::cout << "CPU network" << std::endl;
			net.print_network();
			std::cout << "GPU network" << std::endl;
			print_net();
			std::cout << "Validates: " << validate_weights(net.wInputHidden, net.wHiddenOutput) << std::endl;
			std::getline(std::cin, r);
			if (r == "exit") {
				return;
			}
		}
	}
}



size_t GPUNet::current_mem_usage(int dev) {
	return gpu_mem[dev];
}



/*
 * ------------ private ------------
 */


void GPUNet::add_gpu_mem(int bytes) {
	gpu_mem[get_current_device()] += bytes;
}


int GPUNet::get_current_device() {
	int device;
	hipGetDevice(&device);
	return device;
}

size_t GPUNet::dataset_size(TrainingDataSet *tset) {
	size_t tset_size = 0;
	int fv_size = (n_input + n_output) * sizeof(float);
	tset_size += fv_size * tset->training_set.size();
	tset_size += fv_size * tset->generalization_set.size();
	tset_size += fv_size * tset->validation_set.size();
	return tset_size;
}

size_t GPUNet::total_dev_mem(int dev) {
	hipDeviceProp_t props;
	hipGetDeviceProperties(&props, dev);
	return props.totalGlobalMem - 1611000000; //minus 1.5 gb
}



/*
 * Copies the host vector to a pointer array on the host that holds pointers to FeatureVector on the device with bias node
 */
void GPUNet::copy_to_device_host_array_ptrs_biased(thrust::host_vector<FeatureVector*> &hv, FeatureVector ***dv) {
	std::cout << "Copying data" << std::endl;

	*dv = (FeatureVector**)malloc(hv.size()*sizeof(FeatureVector*));
	//FeatureVector** host_dv_tmp = (FeatureVector**)malloc(hv.size()*sizeof(FeatureVector*));

	for (size_t i = 0; i < hv.size(); ++i) {
		//allocate device memory
		FeatureVector *d_fv = (FeatureVector*)malloc(sizeof(FeatureVector*));

		float *d_inp, *d_tar;
		CUDA_CHECK_RETURN(hipMalloc((void **)&d_inp, (n_input+1)*sizeof(float)));
		CUDA_CHECK_RETURN(hipMalloc((void **)&d_tar, (n_output)*sizeof(float)));

		//TODO: cuda-memcheck claims there is an unspecified launch failure at this line...
		CUDA_CHECK_RETURN(hipMemcpy(d_inp, hv[i]->input, n_input*sizeof(float), hipMemcpyHostToDevice));
		CUDA_CHECK_RETURN(hipMemcpy(d_tar, hv[i]->target, n_output*sizeof(float), hipMemcpyHostToDevice));

		d_fv->input = d_inp;
		d_fv->target = d_tar;

		//TODO: does setting all in parallel improve speed?
		set_bias<<<1, 1>>>(n_input, d_inp);
		CUDA_CHECK_RETURN(hipDeviceSynchronize());

		(*dv)[i] = d_fv;
	}

}

/**
 * Copy from pattern p_start to p_end to device
 * only allocate memory if \p allocate is true
 */
void GPUNet::copy_to_device_host_array_ptrs_biased_section(thrust::host_vector<FeatureVector*> &hv, FeatureVector ***dv,
		int p_start, int p_end, bool allocate) {

	std::cout << "Copying data, p_start = " << p_start << ", p_end = " << p_end << ", allocate = " << allocate << std::endl;

	if (allocate) { // if the first epoch and the first section
		*dv = (FeatureVector**)malloc(hv.size()*sizeof(FeatureVector*));
	}

	for (int i = p_start, p = 0; i < p_end; ++i, ++p) {
		if (allocate) {
			//allocate device memory
			FeatureVector *d_fv = (FeatureVector*)malloc(sizeof(FeatureVector*));

			float *d_inp, *d_tar;
			CUDA_CHECK_RETURN(hipMalloc((void **)&d_inp, (n_input+1)*sizeof(float)));
			CUDA_CHECK_RETURN(hipMalloc((void **)&d_tar, (n_output)*sizeof(float)));

			CUDA_CHECK_RETURN(hipMemcpy(d_inp, hv[i]->input, n_input*sizeof(float), hipMemcpyHostToDevice));
			CUDA_CHECK_RETURN(hipMemcpy(d_tar, hv[i]->target, n_output*sizeof(float), hipMemcpyHostToDevice));

			d_fv->input = d_inp;
			d_fv->target = d_tar;

			//NOTE: no need to synchronize since on default stream and
			//next GPU function could not start until this one finishes
			set_bias<<<1, 1>>>(n_input, d_inp);

			(*dv)[p] = d_fv;
		} else {
			CUDA_CHECK_RETURN(hipMemcpy((*dv)[p]->input, hv[i]->input, n_input*sizeof(float), hipMemcpyHostToDevice));
			CUDA_CHECK_RETURN(hipMemcpy((*dv)[p]->target, hv[i]->target, n_output*sizeof(float), hipMemcpyHostToDevice));
		}
	}
}
